//#include <thrust/device_vector.h>
//#include <thrust/host_vector.h>

#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
struct dev_vec{
int pos;
int size;
int  D_Vec[100];
//int  *D_Vec;
};
__device__ struct dev_vec  create_coll_device( int n){
struct dev_vec t1;
t1.pos=100;
//t1.size=n;
//t1.D_Vec=(int *)malloc(sizeof(int)*100);
t1.size=0;
return t1;
}
__device__ int get_elem_coll_device(struct dev_vec &t1,int &val){
t1.size--;
if(t1.size<=98 && t1.size>=0){ val=t1.D_Vec[t1.size];return val;}
else{ return 0;}
}
__device__ int add_elem_coll_device(struct dev_vec t1,int pos,int val){
if(t1.size>pos){ t1.D_Vec[t1.pos]=val;return 1;}
if(t1.size==pos && pos<t1.pos){
t1.D_Vec[pos]=val;
t1.size++;
return  1;
}
return 0;
}
__device__ void add_coll_device(struct dev_vec &t1,int value){
if(t1.size>t1.pos){
//printf("EXCESS\n");
//t1.size--;
return;
}
t1.D_Vec[t1.size]=value;
t1.size++;
}
__global__ void  test (){
struct dev_vec t1=create_coll_device(100);
for(int i=0;i<54;i++){
add_coll_device(t1,i*20);
}
int i;
while(t1.size>0){printf("%d ",get_elem_coll_device(t1,i));if((t1.size%10)==0)printf("\n");}
add_elem_coll_device(t1,250,245);

}

/*main(){
test<<<1,1>>>();
cudaDeviceSynchronize();
}*/



