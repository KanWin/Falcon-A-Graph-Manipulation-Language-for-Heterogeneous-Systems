#include "hip/hip_runtime.h"
#ifndef GGRAPH_H
#define GGRAPH_H
#include "Graph.h"
#include <stdio.h>
#include </usr/local/cuda/include/hip/hip_runtime.h>
    #include </usr/local/cuda/include/hip/hip_runtime_api.h>
class HGraph;
class GGraph: public Graph {
public:
//int *maxnpoints,*pnpoints,*maxnedges,*pnedges;
int maxnpoints,*pnpoints,maxnedges,*pnedges;//made this as only pnpoints and pnedges are used in atomic.
int allocate();
 int copytoCPU(HGraph &hgraph);
int  copyfromCPU(HGraph &hgraph);
 int cloneCPU(HGraph &hgraph);
 int deallocate();
//__device__  int addPoint(int x,int y);
//__constant__ union float_int   *edges;
__device__ int getEdgeWeight(int t1,int t2);
__device__ int addPoint(float x,float y);
__device__ int addPoint(int x,int y);
__device__ int addEdge(int p1,int p2);
__device__ int addEdge(int p1,int p2,int weight);
 void (*extra_alloc)(GGraph &);
};
__device__ int GGraph::addPoint(float x,float y){
int x1=1;
int t1;
t1=atomicAdd(pnpoints,x1);
//if(t1<*(maxnpoints)){
if(t1<(maxnpoints)){
points[2*t1].fpe=x;
points[2*t1+1].fpe=y;
}
return t1;
}
__device__ int GGraph::addPoint(int x,int y){
int x1=1;
int t1;
t1=atomicAdd(pnpoints,x1);
//if(t1<*(maxnpoints)){
if(t1<(maxnpoints)){
points[2*t1].ipe=x;
points[2*t1+1].ipe=y;
}
return t1;
}
__device__ int GGraph::addEdge(int p1,int p2){
int x=1;
int t1;
t1=atomicAdd(pnedges,x);
//if(t1<*(maxnpoints)){
if(t1<(maxnpoints)){
edges[3*t1].ipe=p1;
edges[3*t1+1].ipe=p2;
edges[3*t1+2].ipe=0;

}
return t1;
}
__device__ int GGraph::addEdge(int p1,int p2,int weight){
int x=1;
int t1;
t1=atomicAdd(pnedges,x);
//if(t1<*(maxnpoints)){
if(t1<(maxnpoints)){
edges[3*t1].ipe=p1;
edges[3*t1+1].ipe=p2;
edges[3*t1+2].ipe=weight;
}
return t1;
}
//__device__
 volatile unsigned int *arrayin,*arrayout;
__device__ volatile unsigned int ARRU[1024],ARRO[1024];
unsigned int goal;
void alloc_sync_array(){
	if (hipMalloc((void **)&arrayin, 1024*sizeof(volatile unsigned int )) != hipSuccess) printf("allocating arrayin failed");
	if (hipMalloc((void **)&arrayout, 1024*sizeof(volatile unsigned int )) != hipSuccess) printf("allocating arrayout failed");
}
__device__ void __gpu_sync(unsigned goalVal, volatile unsigned *Arrayin, volatile unsigned *Arrayout) {
	// thread ID in a block
	unsigned tid_in_blk = threadIdx.x * blockDim.y + threadIdx.y;
	unsigned nBlockNum = gridDim.x * gridDim.y;
	unsigned bid = blockIdx.x * gridDim.y + blockIdx.y;
	// only thread 0 is used for synchronization
	if (tid_in_blk == 0) {
		Arrayin[bid] = goalVal;
		__threadfence();
	}
	if (bid == 0) {
		if (tid_in_blk < nBlockNum) {
			while (Arrayin[tid_in_blk] != goalVal){
				//Do nothing here
			}
		}
		__syncthreads();
		if (tid_in_blk < nBlockNum) {
			Arrayout[tid_in_blk] = goalVal;
			__threadfence();
		}
	}
	if (tid_in_blk == 0) {
		while (Arrayout[bid] != goalVal) {
			//Do nothing here
		}
	}
	__syncthreads();
}
/*

__device__ void __gpu_sync(unsigned goalVal, volatile unsigned *Arrayin, volatile unsigned *Arrayout) {
	// thread ID in a block
	unsigned tid_in_blk = threadIdx.x ;//
	unsigned nBlock = gridDim.x ;//0
	unsigned bid = blockIdx.x  ;//0

   if(bid==0){
	if (tid_in_blk == 0) {
		Arrayin[bid] = 1;
		__threadfence();
	}
		__threadfence();
__syncthreads();
for(int pb=tid_in_blk;pb<nBlock;pb+=1){
 while(Arrayin[pb]==0){
__threadfence_block();
}
}
__syncthreads();

		__threadfence();
 
for(int pb=tid_in_blk;pb<nBlock;pb+=1){

		Arrayin[pb] =0;
		__threadfence();
}

}
else {

if(tid_in_blk==0){
Arrayin[bid]=1;

		__threadfence();
 while(Arrayin[bid]==1){
__threadfence_block();
}
}
__syncthreads();
}
}*/
#endif
