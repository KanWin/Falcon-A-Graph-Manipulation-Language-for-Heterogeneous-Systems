#include "hip/hip_runtime.h"

 #include "dynamicsssp.h"
hipDeviceProp_t prop0;
__device__ int   changed =0;
;
 int   hchanged =0;


 void   processincrementation ( char    *  name ,HGraph & hgraph ) 
 {

 FILE*fp1; 

 fp1=fopen(name,"r");


  int   src , dst , weight ;


  int   i =0;


 
 while(fscanf(fp1,"%d%d%d",&src,&dst,&weight)!=EOF)  { 

 hgraph.addEdge( src,dst,weight); 

 ((struct struct_hgraph  *)(hgraph.extra))->updated[src]=true; 

 i++; 

 }

 }
__global__ void   relaxgraph ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < graph.npoints&& ((struct struct_hgraph  *)(graph.extra))->/*4*/updated[id]==true ){

 ((struct struct_hgraph  *)(graph.extra))->updated[id]=false; 

 int falcft0=graph.outtotal[id];
int falcft1=graph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
int t=ut1;
GMIN(&(((struct struct_hgraph  *)(graph.extra))->dist[ut1]),((struct struct_hgraph  *)(graph.extra))->dist[id]+ut2,changed);
}
}

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 ((struct struct_hgraph  *)(graph.extra))->olddist[id]=1234567890; 

 ((struct struct_hgraph  *)(graph.extra))->updated[id]=false; 

 }

}
__global__ void   reset1 ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 if( ((struct struct_hgraph  *)(graph.extra))->dist[id]<((struct struct_hgraph  *)(graph.extra))->olddist[id] )
((struct struct_hgraph  *)(graph.extra))->updated[id]=true; 

 ((struct struct_hgraph  *)(graph.extra))->olddist[id]=((struct struct_hgraph  *)(graph.extra))->dist[id]; 

 }

}
void   SSSP ( char    *  name ,char    *  name1 ) 
 {

 HGraph  hgraph ;

 

 

 GGraph graph;


 

 hgraph.readMorph(name);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


  double   t1 , t2 ;


 t1=rtclock();


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 bool  falcvt1;
falcvt1=true;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.updated[0]),&(falcvt1),sizeof(bool),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 6");

 int   falcvt2;
falcvt2=0;
struct struct_hgraph  temp2;
 hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp2.dist[0]),&(falcvt2),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 7");

 
 while(1)  { 

 int   falcvt3;
falcvt3=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt3),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 8");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
relaxgraph<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int   falcvt4;
if(hipMemcpyFromSymbol(&(falcvt4),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 9");
if(falcvt4==0)break;

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset1<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 }

 struct struct_hgraph  temp3;/*err*/
hipMemcpy(&temp3,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp3.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 10");

 t2=rtclock();


 printf("\nTIME =%f",(t2-t1)*1000);


  int   maxdist =0;


 for (int   i =0;i<hgraph.npoints;i++) {

 if( maxdist<((struct struct_hgraph  *)(hgraph.extra))->dist[i]  )
maxdist=((struct struct_hgraph  *)(hgraph.extra))->dist[i]; 

 }

 printf("\nMAXDIST=%d\n",maxdist);


 processincrementation(name1,hgraph);


 struct struct_hgraph  temp4;/*err*/
hipMemcpy(&temp4,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);

//updated updated
//updated dist
if(hipMemcpy((temp4.updated),(((struct struct_hgraph  *)(hgraph.extra))->updated),sizeof(bool)*graph.npoints,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 11");

 hgraph.copytoGPU(graph);hipSetDevice(0);


 
 while(1)  { 

 int   falcvt5;
falcvt5=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt5),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 12");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
relaxgraph<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int   falcvt6;
if(hipMemcpyFromSymbol(&(falcvt6),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 13");
if(falcvt6==0)break;

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset1<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 }

 struct struct_hgraph  temp5;/*err*/
hipMemcpy(&temp5,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp5.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 14");

 maxdist=0; 

 for (int   i =0;i<hgraph.npoints;i++) {

 if( maxdist<((struct struct_hgraph  *)(hgraph.extra))->dist[i]  )
maxdist=((struct struct_hgraph  *)(hgraph.extra))->dist[i]; 

 }

 printf("\nMAXDIST=%d\n",maxdist);


 return ;

 }
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


 if( argc!=3  )
{

 printf("error:-exec   file1 file2");


 return 1;

 }

 SSSP(argv[1],argv[2]);


 }
