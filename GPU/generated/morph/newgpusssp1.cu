#include "hip/hip_runtime.h"

 #include "newgpusssp1.h"
hipDeviceProp_t prop0;
__device__ int   changed =0;
;
 int   hchanged =0;


 void   processincrementation ( HGraph & hgraph ,char    *  name ) 
 {

 fopen(name,"r");


  int   src , dst , weight ;


  int   i =0;


 
 while(fscanf(fp1,"%d%d%d",&src,&dst,&weight)!=EOF)  { 

 i++; 

 hgraph.addEdge( src,dst,weight); 

 }

 }
__global__ void   relaxgraph ( GGraph  unnigraph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int p=id;
if( id < unnigraph.npoints&& ((struct struct_hgraph  *)(unnigraph.extra))->/*4*/updated[id]==true ){

 ((struct struct_hgraph  *)(unnigraph.extra))->updated[id]=false; 

 int falcft0=unnigraph.index[id+1]-unnigraph.index[id];
int falcft1=unnigraph.index[id];
for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=unnigraph.edges[ut0].ipe;
int ut2=unnigraph.edges[ut0+1].ipe;
int t=ut1;
GMIN(&(((struct struct_hgraph  *)(unnigraph.extra))->dist[ut1]),((struct struct_hgraph  *)(unnigraph.extra))->dist[id]+ut2,changed);
}
}

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 ((struct struct_hgraph  *)(graph.extra))->olddist[id]=1234567890; 

 ((struct struct_hgraph  *)(graph.extra))->updated[id]=false; 

 }

}
__global__ void   reset1 ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if( id < graph.npoints){

 if( ((struct struct_hgraph  *)(graph.extra))->dist[id]<((struct struct_hgraph  *)(graph.extra))->olddist[id] )
((struct struct_hgraph  *)(graph.extra))->updated[id]=true; 

 ((struct struct_hgraph  *)(graph.extra))->olddist[id]=((struct struct_hgraph  *)(graph.extra))->dist[id]; 

 }

}
void   SSSP ( char    *  name ,char    *  name2 ) 
 {

 HGraph  hgraph ;

 

 GGraph graph;


 

 

 hgraph.readmorph(name);

int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 //val==1 
hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


  double   t1 , t2 ;


 t1=rtclock();


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 //val==1 
bool  falcvt1;
falcvt1=true;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.updated[0]),&(falcvt1),sizeof(bool),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 6");

 //val==1 
int   falcvt2;
falcvt2=0;
struct struct_hgraph  temp2;
 hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp2.dist[0]),&(falcvt2),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 7");

 
 while(1)  { 

 //val==1 
int   falcvt3;
falcvt3=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt3),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 8");

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
relaxgraph<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 int   falcvt4;
if(hipMemcpyFromSymbol(&(falcvt4),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 9");
if(falcvt4==0)break;

 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset1<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);


 }

 processincrementation(hgraph,name2);


 t2=rtclock();


 printf("\nTIME =%f",(t2-t1)*1000);


 //val==2 
struct struct_hgraph  temp3;/*err*/
hipMemcpy(&temp3,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
//dist dist
//dist 
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp3.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 10");

  int   maxdist =0;


 for (int   i =0;i<hgraph.npoints;i++) {

 if( maxdist<((struct struct_hgraph  *)(hgraph.extra))->dist[i]  )
maxdist=((struct struct_hgraph  *)(hgraph.extra))->dist[i]; 

 }

 printf("\nMAXDIST=%d\n",maxdist);


 return ;

 }
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


 if( argc!=2  )
{

 printf("error: exec filename");


 return 1;

 }

 SSSP(argv[1],argv[2]);


 }
