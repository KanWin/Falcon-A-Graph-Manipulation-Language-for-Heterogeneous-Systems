#include "hip/hip_runtime.h"

 #include "falcondmr.h"
hipDeviceProp_t prop0;
 int   ntria ;


 
#define PI 3.141593

 __device__ int   changed , pnnodes , pntriangles , success , badcount ;
;
 int   hchanged , hbadcount ;


 __device__ int   aborted ;
;


 

 

 HGraph  hgraph ;

 void   fun ( ) 
 {

 

 }
void   readTriangle ( HGraph & hgraph ,char    *  name ) 
 {

 FILE*fp; 

 FILE*fp1; 

  char   temp [ 100] ;


  int   ntriaone , ntriatwo , n1 , n2 , n3 , n4 ;


  char    *  name1 ,  *  name2 ;


 name1=(char *)malloc(sizeof(char )*100); 

 name2=(char *)malloc(sizeof(char )*100); 

 sprintf(name1,"%s.ele",name);


 sprintf(name2,"%s.poly",name);


 fp=fopen(name1,"r");


 fscanf(fp,"%d",&ntriaone);


 fgets(temp,100,fp);


 for (int   i =0;i<ntriaone;i++) {

 fscanf(fp,"%d%d%d%d",&n1,&n2,&n3,&n4);


 ((struct struct_hgraph  *)(hgraph.extra))->triangle[n1].nodes[0]=n2; 

 ((struct struct_hgraph  *)(hgraph.extra))->triangle[n1].nodes[1]=n3; 

 ((struct struct_hgraph  *)(hgraph.extra))->triangle[n1].nodes[2]=n4; 

 fgets(temp,100,fp);


 }

 fclose(fp);


 fp1=fopen(name2,"r");


 fgets(temp,100,fp1);


 fscanf(fp1,"%d",&ntriatwo);


 fgets(temp,100,fp1);


 for (int   i =0;i<ntriatwo;i++) {

 fscanf(fp1,"%d%d%d%d",&n1,&n2,&n3,&n4);


 ((struct struct_hgraph  *)(hgraph.extra))->triangle[n1+ntriaone].nodes[0]=n2; 

 ((struct struct_hgraph  *)(hgraph.extra))->triangle[n1+ntriaone].nodes[1]=n3; 

 ((struct struct_hgraph  *)(hgraph.extra))->triangle[n1+ntriaone].nodes[2]=INVALID; 

 fgets(temp,100,fp1);


 }

 fclose(fp1);


 }
void   readnbrsfromfile ( char    *  name ,HGraph & hgraph ) 
 {

  char    *  filename =(char *)malloc(sizeof(char )*100);


 sprintf(filename,"%s.nbr",name);


 FILE*fp2; 

 fp2=fopen(filename,"r");


 if( fp2==NULL  )
printf("********OPEN FALIED\n");


  int   ttt ;


 for (int   i =0;i<((struct struct_hgraph  *)(hgraph.extra))->ntriangle;i++) {

 fscanf(fp2,"%d",&ttt);


 fscanf(fp2,"%d%d%d",&(((struct struct_hgraph  *)(hgraph.extra))->triangle[i].neighbours[0]),&(((struct struct_hgraph  *)(hgraph.extra))->triangle[i].neighbours[1]),&(((struct struct_hgraph  *)(hgraph.extra))->triangle[i].neighbours[2]));


 fscanf(fp2,"%d%d%d",&(((struct struct_hgraph  *)(hgraph.extra))->triangle[i].neighedgestart[0]),&(((struct struct_hgraph  *)(hgraph.extra))->triangle[i].neighedgestart[1]),&(((struct struct_hgraph  *)(hgraph.extra))->triangle[i].neighedgestart[2]));


 }

 fclose(fp2);


 }
__device__ int   addTriangle ( int   p1 ,int   p2 ,int   p3 ,GGraph  graph ) 
 {

  int   t1 ;


 int  tr1 ;


 t1=GADD(&(devngraphtriangle),1);


 ((struct struct_hgraph  *)(graph.extra))->triangle[t1].nodes[0]=p1; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[t1].nodes[1]=p2; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[t1].nodes[2]=p3; 

 // this is a access where rhs needs to b converted to int 
tr1=t1;


 return t1;

 }
__device__ int   addTriangle1 ( int   p1 ,int   p2 ,GGraph  graph ) 
 {

  int   t1 ;


 int  tr1 ;


 t1=GADD(&(devngraphtriangle),1);


 ((struct struct_hgraph  *)(graph.extra))->triangle[t1].nodes[0]=p1; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[t1].nodes[1]=p2; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[t1].nodes[2]=INVALID; 

 // this is a access where rhs needs to b converted to int 
tr1=t1;


 return t1;

 }
__global__ void   verify ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int T=id;
if(id <(( struct struct_hgraph  *)(graph.extra))->ntriangle){

  int   x ;


 if( ((struct struct_hgraph  *)(graph.extra))->triangle[id].isdel==0&&((struct struct_hgraph  *)(graph.extra))->triangle[id].isbad==1 )
{
 changed=1; 
 }

 }

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if(id <(( struct struct_hgraph  *)(graph.extra))->ntriangle){

 ((struct struct_hgraph  *)(graph.extra))->triangle[id].owner=1234567890; 

 }

}
__device__ float   distanceSquare ( struct Point  p1 ,struct Point  p2 ,GGraph  graph ) 
 {

  float   x1 =p1.x-p2.x;


  float   y1 =p1.y-p2.y;


  float   dsqr =x1*x1+y1*y1;


 return dsqr;

 }
__device__ float   distance ( struct Point  one ,struct Point  two ,GGraph  graph ) 
 {

 return sqrtf(distanceSquare(one,two,graph));

 }
__device__ int   getOpposite ( int   t1 ,GGraph  graph ,int   obtuse ) 
 {

  int   dims ;


 dims=(((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].nodes[2]==INVALID) ?2 : 3; 

  int   commonedgepoint1 =(obtuse+1)%dims;


 for (unsigned   ii =0;ii<dims;++ii) {

  int   nnedgestart =((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].neighedgestart[ii];


 if( nnedgestart==commonedgepoint1 )
{

 int  pp ;


 // this is a access where rhs needs to b converted to int 
pp=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].neighedgestart[ii]; 

 return pp;

 }

 }

 return INVALID;

 }
__device__ bool  copyNeighbours ( int   t1 ,int   t2 ,GGraph  graph ) 
 {

  int   dims ;


 for (int   i =0;i<3;i++) {

 ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].neighbours[i]=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t2].neighbours[i]; 

 ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].neighedgestart[i]=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t2].neighedgestart[i]; 

 }

 return false;

 }
__device__ bool  updateNeighbour ( int   t1 ,int   t2 ,int   t3 ,GGraph  graph ) 
 {

  int   dims ;


 dims=(((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].nodes[2]==INVALID) ?2 : 3; 

 for (int   i =0;i<dims;i++) {

 if( ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].neighbours[i]==t2 )
{

 ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].neighbours[i]=t3; 

 }

 }

 return false;

 }
__device__ bool  checkbad ( int   t ,GGraph  graph ) 
 {

 for (int   i =0;i<3;i++) {

 struct Point p1,p2,p3;

  int   te =(i+1)%3;


  int   te2 =(i+2)%3;


 if( ((struct struct_hgraph  *)(graph.extra))->triangle[t].nodes[i]!=INVALID&&((struct struct_hgraph  *)(graph.extra))->triangle[t].nodes[te]!=INVALID&&((struct struct_hgraph  *)(graph.extra))->triangle[t].nodes[te2]!=INVALID )
{

 int repli0=((struct struct_hgraph  *)(graph.extra))->triangle[t].nodes[i];
p1.x=graph.points[2*repli0].fpe;
p1.y=graph.points[2*repli0+1].fpe;


 int repli1=((struct struct_hgraph  *)(graph.extra))->triangle[t].nodes[te];
p2.x=graph.points[2*repli1].fpe;
p2.y=graph.points[2*repli1+1].fpe;


 int repli2=((struct struct_hgraph  *)(graph.extra))->triangle[t].nodes[te2];
p3.x=graph.points[2*repli2].fpe;
p3.y=graph.points[2*repli2+1].fpe;


  float   vax =p2.x-p1.x;


  float   vay =p2.y-p1.y;


  float   vbx =p3.x-p1.x;


  float   vby =p3.y-p1.y;


  float   dp =vax*vbx+vay*vby;


 if( dp<0 )
((struct struct_hgraph  *)(graph.extra))->triangle[t].obtuse=i; 
else {

  float   dstsqrap1 =distanceSquare(p2,p1,graph);


  float   dstsqrap2 =distanceSquare(p3,p1,graph);


  float   c =dp*rsqrtf(dstsqrap1*dstsqrap2);


 if( c>cos(30.000000*PI/180.000000) )
{

 atomicAdd(&badcount,1);


 return 1;

 }

 }

 }

 }

 return 0;

 }
__device__ int   adjacent ( int   t1 ,int   t2 ,GGraph  graph ) 
 {

  int   common , firstmatch ;


 common=0; 

 firstmatch=3; 

  int   dimone , dimtwo ;


 if( ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].nodes[2]==INVALID )
dimone=2; 
else dimone=3; 

 if( ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t2].nodes[2]==INVALID )
dimtwo=2; 
else dimtwo=3; 

 for (int   i =0;i<dimone;i++) {

 for (int   j =0;j<dimtwo;j++) {

 if( ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t1].nodes[i]==((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t2].nodes[j] )
{

 if( ++common==2 )
return firstmatch;
else firstmatch=i; 

 }

 }

 }

 return 3;

 }
__global__ void   dinit ( GGraph  graph ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;
if(id <(( struct struct_hgraph  *)(graph.extra))->ntriangle){

 ((struct struct_hgraph  *)(graph.extra))->triangle[id].obtuse=3; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[id].isbad=checkbad(id,graph);


 ((struct struct_hgraph  *)(graph.extra))->triangle[id].isdel=0; 

 devngraphtriangle=((struct struct_hgraph  *)(graph.extra))->ntriangle; 

 }

}
__device__ struct  cent  getCenter ( int   t ,GGraph  graph ) 
 {

   struct cent   ce1 ;


 struct Point p1,p2,p3;

  int   dims ;


 if( ((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[2]==INVALID )
dims=2; 
else dims=3; 

 int repli3=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[0];
p1.x=graph.points[2*repli3].fpe;
p1.y=graph.points[2*repli3+1].fpe;


 int repli4=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[1];
p2.x=graph.points[2*repli4].fpe;
p2.y=graph.points[2*repli4+1].fpe;


 if( dims==3 )
{
 int repli5=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[2];
p3.x=graph.points[2*repli5].fpe;
p3.y=graph.points[2*repli5+1].fpe;

 }

 if( !(((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[0]<((struct struct_hgraph  *)(graph.extra))->ntriangle&&((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[1]<((struct struct_hgraph  *)(graph.extra))->ntriangle&&((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[2]<((struct struct_hgraph  *)(graph.extra))->ntriangle) )
{

 ce1.centerx=0.000000; 

 ce1.centery=0.000000; 

 return ce1;

 }

 if( dims==2 )
{

 ce1.centerx=(p1.x+p2.x)*0.500000; 

 ce1.centery=p1.y+p1.y*0.500000; 

 return ce1;

 }

  float   vax =p2.x-p1.x;


  float   vay =p2.y-p1.y;


  float   vbx =p3.x-p1.x;


  float   vby =p3.y-p1.y;


  float   xxlen =distance(p1,p2,graph);


  float   yylen =distance(p1,p3,graph);


  float   cosine =(vax*vbx+vay*vby)/(xxlen*yylen);


  float   sinesq =1.000000-cosine*cosine;


  float   plen =yylen/xxlen;


  float   ss =plen*cosine;


  float   tt =plen*sinesq;


  float   wp =(plen-cosine)/(2.000000*tt);


  float   wb =0.500000-(wp*ss);


 ce1.centerx=p1.x*(1.000000-wb-wp)+p2.x*wb+p3.x*wp; 

 ce1.centery=p1.y*(1.000000-wb-wp)+p2.y*wb+p3.y*wp; 

 return ce1;

 }
__device__ float   radiusSqaure ( struct Point  p1 ,int   t ,GGraph  graph ) 
 {

 struct Point p2;

 int repli6=((struct struct_hgraph  *)(graph.extra))->/*4*/triangle[t].nodes[0];
p2.x=graph.points[2*repli6].fpe;
p2.y=graph.points[2*repli6+1].fpe;


 return distanceSquare(p1,p2,graph);

 }
__device__ bool  inCircumCircle ( struct Point  P0 ,int   t1 ,GGraph  graph ) 
 {

   struct cent   ce1 ;


 ce1=getCenter(t1,graph);


 struct Point P1,P2;

 P1.x=ce1.centerx; 

 P1.y=ce1.centery; 

  float   dd =distanceSquare(P1,P0,graph);


 return dd<=radiusSqaure(P1,t1,graph);

 }
__global__ void   drefine ( GGraph  graph ,int   nblocks ,int   size ,int   goal ,unsigned volatile int *   arrayin ,unsigned volatile int *   arrayout ,int FALCX) 
 {
int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
 
int t=id;

   struct cent   cent_new ;


 struct dev_vec pred,frontier,post,conntria1,conntria2;
pred= create_coll_device(100);
frontier= create_coll_device(100);
post= create_coll_device(100);
conntria1= create_coll_device(100);
conntria2= create_coll_device(100);


 struct dev_vec conn1,conn2,conn3,conn4;
conn1= create_coll_device(100);
conn2= create_coll_device(100);
conn3= create_coll_device(100);
conn4= create_coll_device(100);


 int  ceelem ;


  int   lchanged =0;


  int   successfull =0;


 unsigned nthreads=blockDim.x*gridDim.x;
 unsigned wpt=(((struct struct_hgraph  *)(graph.extra))->ntriangle+nthreads)/nthreads;
unsigned start=id*wpt;
unsigned end=start+wpt;
 for(unsigned id=start;id<end;id++){
if(id < (( struct struct_hgraph  *)(graph.extra))->ntriangle){
if( !(((struct struct_hgraph  *)(graph.extra))->triangle[id].isdel)&&(((struct struct_hgraph  *)(graph.extra))->triangle[id].isbad) )
{

 ceelem=id; 

  int   cebot ;


 cebot=((struct struct_hgraph  *)(graph.extra))->triangle[id].obtuse; 

  int   itr =0;


 
 while(cebot<3&&++itr<50&&ceelem<((struct struct_hgraph  *)(graph.extra))->ntriangle)  { 

  int   flag ;


 flag=getOpposite(ceelem,graph,cebot);


 if( flag<((struct struct_hgraph  *)(graph.extra))->ntriangle )
{

 // this is a access where rhs needs to b converted to int 
ceelem=flag;


 cebot=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].obtuse; 

 }

 }

 if( ceelem>=((struct struct_hgraph  *)(graph.extra))->ntriangle||((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].isdel )
{

 ceelem=id; 

 cebot=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].obtuse; 

 }

 cent_new=getCenter(ceelem,graph);


 struct Point P1;

 int P0;

 P1.x=cent_new.centerx; 

 P1.y=cent_new.centery; 

 add_coll_device(pred,ceelem); 

 add_coll_device(frontier,ceelem); 

 
 while(frontier.size>0)  { 

 int  curr ;


 get_elem_coll_device(frontier,curr); 

 if( curr>=((struct struct_hgraph  *)(graph.extra))->ntriangle )
break;

  int   dims ;


 dims=(((struct struct_hgraph  *)(graph.extra))->triangle[curr].nodes[2]==INVALID) ?2 : 3; 

 for (int   i =0;i<dims;i++) {

 int  next ;


 // this is a access where rhs needs to b converted to int 
next=((struct struct_hgraph  *)(graph.extra))->triangle[curr].neighbours[i]; 

 if( next>=((struct struct_hgraph  *)(graph.extra))->ntriangle )
break;

 if( next>=((struct struct_hgraph  *)(graph.extra))->ntriangle )
break;

 if( ((struct struct_hgraph  *)(graph.extra))->triangle[next].isdel )
continue;

  int   nextdims ;


 nextdims=(((struct struct_hgraph  *)(graph.extra))->triangle[next].nodes[2]==INVALID) ?2 : 3; 

 if( !(next!=ceelem&&dims==2&&nextdims==2)&&inCircumCircle(P1,next,graph) )
{

 if( nextdims==2&&dims!=2 )
{

 frontier.size=0; 

 pred.size=0; 

 conn1.size=0; 

 conn2.size=0; 

 conntria1.size=0; 

 conntria2.size=0; 

 ceelem=next; 

 cebot=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].obtuse; 

 itr=0; 

 
 while(cebot<3&&++itr<50&&ceelem<((struct struct_hgraph  *)(graph.extra))->ntriangle)  { 

  int   flag ;


 flag=getOpposite(ceelem,graph,cebot);


 if( flag<((struct struct_hgraph  *)(graph.extra))->ntriangle )
{

 // this is a access where rhs needs to b converted to int 
ceelem=flag;


 cebot=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].obtuse; 

 }

 }

 if( ceelem>=((struct struct_hgraph  *)(graph.extra))->ntriangle||((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].isdel )
{

 ceelem=next; 

 cebot=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].obtuse; 

 }

 cent_new=getCenter(ceelem,graph);


 P1.x=cent_new.centerx; 

 P1.y=cent_new.centery; 

 add_coll_device(pred,ceelem); 

 add_coll_device(frontier,ceelem); 

 }
else {
  unsigned   jj ;


 for (jj=0;jj<pred.size;jj++) {

 if( pred.D_Vec[jj]==next )
break;

 }

 if( jj==pred.size )
{

 add_coll_device(pred,next); 

 add_coll_device(frontier,next); 

 }

 }

 }
else {

  int   pi1 =((struct struct_hgraph  *)(graph.extra))->triangle[curr].neighedgestart[i];


 if( pi1>=3 )
continue;

 int p1=((struct struct_hgraph  *)(graph.extra))->triangle[curr].nodes[pi1];

 int p2=((struct struct_hgraph  *)(graph.extra))->triangle[curr].nodes[(pi1+1)%3];

  unsigned   jj ;


 for (jj=0;jj<conn1.size;jj+=1) {

 if( conn1.D_Vec[jj]==p1&&conn2.D_Vec[jj]==p2 )
break;

 }

 if( jj==conn1.size )
{

 add_coll_device(conn1,p1); 

 add_coll_device(conn2,p2); 

 add_coll_device(conntria1,curr); 

 add_coll_device(conntria2,next); 

 }

 }

 }

 }

 }

 }
bool backoff=false;
for(int i=0;i<pred.size;i++)
 	 ((struct struct_hgraph  *)(graph.extra))->owner[pred.D_Vec[i]]=id;
__gpu_sync(++goal,arrayin,arrayout);
for(int i=0;i<pred.size;i++){
 	 if(( (struct struct_hgraph  *   )(graph.extra))->owner[pred.D_Vec[i]]<id){backoff=true;break;}
 	 else if(((struct struct_hgraph  *)(graph.extra))->owner[pred.D_Vec[i]]>id)((struct struct_hgraph  *)(graph.extra))->owner[pred.D_Vec[i]]=id; 
}
__gpu_sync(++goal,arrayin,arrayout);
if(backoff==0){
 	 for(int i=0;i<pred.size;i++){
 	 	if(( (struct struct_hgraph  *   )(graph.extra))->owner[pred.D_Vec[i]]!=id){backoff=true;break;
 	 	}
 	}
}
if(id < (( struct struct_hgraph  *)(graph.extra))->ntriangle){
if(backoff==0)  {

 if( !(((struct struct_hgraph  *)(graph.extra))->triangle[id].isdel)&&(((struct struct_hgraph  *)(graph.extra))->triangle[id].isbad) )
{

 ++successfull; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[id].isdel=1; 

 int p0;

  int   dims ;


 dims=(((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].nodes[2]==INVALID) ?2 : 3; 

 p0=graph.addPoint(cent_new.centerx,cent_new.centery); 

 int inv_point;

 if( dims==2 )
{

 int  tr1 , tr2 ;


 inv_point=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].nodes[0]; 

 tr1=addTriangle1(p0,inv_point,graph);


 inv_point=((struct struct_hgraph  *)(graph.extra))->triangle[ceelem].nodes[1]; 

  int   octtt ;


 octtt=addTriangle1(p0,inv_point,graph);


 // this is a access where rhs needs to b converted to int 
tr2=octtt;


 add_coll_device(post,tr1); 

 add_coll_device(post,tr2); 

 copyNeighbours(tr1,ceelem,graph);


 copyNeighbours(tr2,ceelem,graph);


 }

 for (unsigned   ii =0;ii<conn1.size;ii+=1) {

 int p1=conn1.D_Vec[ii];

 int p2=conn2.D_Vec[ii];

 int  connsrc ;


 connsrc=conntria1.D_Vec[ii]; 

 int  conndst ;


 conndst=conntria2.D_Vec[ii]; 

 int  newtri ;


  int   octtt1 ;


 octtt1=addTriangle(p0,p1,p2,graph);


 // this is a access where rhs needs to b converted to int 
newtri=octtt1;


  int   jj1 ;


 for (jj1=0;jj1<pred.size;jj1=jj1+1) {

 if( pred.D_Vec[jj1]==conndst )
break;

 jj1++; 

 }

 int  newconn ;


 if( jj1==pred.size )
newconn=conndst; 
else newconn=connsrc; 

  int   iinbr =0;


 ((struct struct_hgraph  *)(graph.extra))->triangle[newtri].neighbours[0]=newconn; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[newtri].neighedgestart[0]=1; 

 iinbr++; 

 for (unsigned   jj =0;jj<post.size;++jj) {

 int  tppp ;


 tppp=post.D_Vec[jj]; 

  int   commonedgestart =adjacent(tppp,newtri,graph);


 if( commonedgestart<3&&iinbr<3 )
{

 ((struct struct_hgraph  *)(graph.extra))->triangle[newtri].neighbours[iinbr]=post.D_Vec[jj]; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[newtri].neighedgestart[iinbr]=commonedgestart; 

 ++iinbr; 

 int  temp1 ;


 temp1=post.D_Vec[jj]; 

 int  unx ;


 unx=post.D_Vec[jj]; 

 updateNeighbour(unx,newconn,newtri,graph);


 }

 }

 add_coll_device(post,newtri); 

 }

  unsigned   jj ;


 for (jj=0;jj<pred.size;++jj) {

 int  temp1 ;


 get_elem_coll_device(pred,temp1); 

 ((struct struct_hgraph  *)(graph.extra))->triangle[temp1].isdel=1; 

 }

 for (unsigned   ii =0;ii<post.size;++ii) {

 int  xyx ;


 get_elem_coll_device(post,xyx); 

 ((struct struct_hgraph  *)(graph.extra))->triangle[xyx].obtuse=3; 

 ((struct struct_hgraph  *)(graph.extra))->triangle[xyx].isbad=checkbad(xyx,graph);


 lchanged|=((struct struct_hgraph  *)(graph.extra))->triangle[xyx].isbad; 

 }

 if( lchanged )
{

 changed=1; 

 }

 }

 }else {

 if( !(((struct struct_hgraph  *)(graph.extra))->triangle[id].isdel)&&(((struct struct_hgraph  *)(graph.extra))->triangle[id].isbad) )
{

 lchanged=1; 

 ++aborted; 

 continue;

 }

 }

 pred.size=0; 

 frontier.size=0; 

 conn1.size=0; 

 post.size=0; 

 conn2.size=0; 

 conntria1.size=0; 

 conntria2.size=0; 

 }

}
}
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();


  int   ntria ;


 fun();


 hgraph.readNodes(argv[1],2); 
int hosthgraph=1;
hgraph.extra=(struct struct_hgraph  *)malloc(sizeof(struct struct_hgraph ));
 read_hgraph_pptysize(hgraph);
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);


 readTriangle(hgraph,argv[1]);


 readnbrsfromfile(argv[1],hgraph);


 GGraph graph;


 hgraph.cloneGPU(graph,0 );
int graphflag=0;
hipSetDevice(0);
hipMalloc((void **)(&graph.extra),sizeof(struct struct_hgraph ));
struct struct_hgraph  temp1;
if(hipMemcpy(&temp1,graph.extra,sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 4");
temp1.ntriangle=(( struct struct_hgraph  *)(hgraph.extra))->ntriangle;
hipMemcpyToSymbol(HIP_SYMBOL(falcgraphtriangle),&((( struct struct_hgraph  *)(hgraph.extra))->ntriangle),sizeof(int),0,hipMemcpyHostToDevice);
if(hipMemcpy(graph.extra,&temp1,sizeof(struct struct_hgraph ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 5");
graphflag=1;
alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);


 struct struct_hgraph  temp2;/*err*/
hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);

//triangle triangle
if(hipMemcpy((temp2.triangle),(((struct struct_hgraph  *)(hgraph.extra))->triangle),sizeof(struct node)*( (    struct struct_hgraph  * )hgraph.extra)->ntriangle,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 6");

 struct struct_hgraph  temp4;
if(hipMemcpy(&temp4,graph.extra,sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 7");
int temp5;
temp5=temp4.ntriangle;
dinit<<<temp5/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,0);
hipDeviceSynchronize();


 if(hipMemcpyFromSymbol(&(hbadcount),badcount,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 8");

 printf("%d \n",hbadcount);


  double   rt1 =rtclock();


 struct struct_hgraph  temp7;
if(hipMemcpy(&temp7,graph.extra,sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 9");
int temp8;
temp8=temp7.ntriangle;
reset<<<temp8/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,0);
hipDeviceSynchronize();


 
 while(1)  { 

 int   falcvt1;
falcvt1=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt1),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 10");

 struct struct_hgraph  temp10;
if(hipMemcpy(&temp10,graph.extra,sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 11");
int temp11;
temp11=temp10.ntriangle;
graph_realloc(graph);
int nblocks,size,goal=0;findthreadandblocksize(prop0,nblocks,size);
resetgraphtriangle<<<temp11/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph ,0 );
hipDeviceSynchronize();
drefine<<<nblocks,size/4>>>(graph,nblocks/***/,size,goal,arrayin,arrayout,0);
hipDeviceSynchronize();
int temp12;
temp12=graph.npoints;
if(hipMemcpy(&(graph.npoints),graph.pnpoints,sizeof(int),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 12");
if(hipMemcpy(graph.pnpoints,&temp12,sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 13");
temp12=graph.nedges;
if(hipMemcpy(&(graph.nedges),graph.pnedges,sizeof(int),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 14");
if(hipMemcpy(graph.pnedges,&temp12,sizeof(int),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 15");
goal=goal+1000;
struct struct_hgraph  temp13;
if(hipMemcpy(&temp13,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 16");
temp12=temp13.ntriangle;
if(hipMemcpyFromSymbol(&(temp13.ntriangle),devngraphtriangle,sizeof(int),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 17");
temp13.pntriangle=temp12;
if(hipMemcpy(((struct struct_hgraph  *)(graph.extra)),&temp13,sizeof(struct struct_hgraph ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 18");


 int   falcvt2;
if(hipMemcpyFromSymbol(&(falcvt2),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 19");
if(falcvt2==0)break;

 }

  double   rt2 =rtclock();


 printf("TIME=%f \n",(rt2-rt1)*1000);


  int   tt ;


 struct struct_hgraph  temp13;/*err*/
hipMemcpy(&temp13,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
// tt ((struct struct_hgraph  *)(graph.extra))->ntriangle 
tt=temp13.ntriangle; 


 printf("%d \n",tt);


 struct struct_hgraph  temp14;/*err*/
int   falcvt3;
falcvt3=tt;
hipMemcpy(&temp14,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
temp14.ntriangle=tt; 


 struct struct_hgraph  temp16;
if(hipMemcpy(&temp16,graph.extra,sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 20");
int temp17;
temp17=temp16.ntriangle;
verify<<<temp17/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,0);
hipDeviceSynchronize();


 }
