#include "hip/hip_runtime.h"

 #include "singlebfs.h"
hipDeviceProp_t prop0;
// clustercomm.c LIB FUNCTION MIN 3rd ARG changed to be communicated
void FALCmpiinit(int argc,char **argv){
MPI_Init(&argc,&argv);
MPI_Comm_rank(MPI_COMM_WORLD, &FALCrank);
MPI_Comm_size(MPI_COMM_WORLD, &FALCsize);
  gethostname(FALChostname,255);
FALCsendbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
FALCrecvbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
FALCsendsize=(int *)malloc(sizeof(int)*FALCsize);
FALCrecvsize=(int *)malloc(sizeof(int)*FALCsize);
for(int i=0;i<FALCsize;i++){
FALCsendsize[i]=FALCrecvsize[i]=0;}
 FALCstatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCrequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
}
__device__ int   changed =0;
;
 int   hchanged ;


 __global__ void   reset ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 }//end fun 0 

}
__global__ void   BFS ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

  int   ch ;


 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
GMIN(&(((struct struct_hgraph  *)(graph.extra))->dist/*xx*/[ut1]),((struct struct_hgraph  *)(graph.extra))->dist[id]+1,/*xx*/changed);//rhs not null


 }//foreach

 }//end fun 0 

}
int   main ( int   argc ,char    *  name [ ] ) 
 {FALCmpiinit(argc,argv);
sprintf(partitionfile,"%s",argv[2]);


 /*s1 0 0*/HGraph  hgraph ;

 

 hgraph.readPointsN(partitionfile,FALCsize);
hgraph.makeNPartitionsMPI(name[1],FALCrank,FALCsize);
int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
 FALCallocbuff(FALCsendbuff,FALCsize,hgraph.remotepoints);
 FALCallocbuff(FALCrecvbuff,FALCsize,hgraph.npoints);
; 

 GGraph graph;


 /*TE=1*///GPU ASS 
hgraph.cloneGPU(graph,0 );
int graphflag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(&prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraphcurrentsize(graph);
hipSetDevice(0);
//val=1 
 


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
reset<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
hipDeviceSynchronize();
hipSetDevice(0);


 /*TE=1*///GPU ASS 
int   falcvt1;
falcvt1=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist[0]),&(falcvt1),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1"); 


 
 while(1)  { 

 /*TE=1*///GPU ASS 
int   falcvt2;
falcvt2=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt2),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 2");//val=1 
 


 hipSetDevice(0);
for(int kk=0;kk<graph.npoints;kk+=graphpointkernelblocks*TPB0){ 
BFS<<<graphpointkernelblocks,TPB0>>>(graph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
hipDeviceSynchronize();
hipSetDevice(0);
for(int kk=1;kk<FALCsize;kk++){
    #pragma omp parallel for   num_threads(32)
    for(int i=graph.offset[kk-1];i<graph.offset[kk];i++){
        sendbuff(i,graph,FALCsendsize,FALCsendbuff,kk-1);
    }
}
for(int i=0;i<FALCsize;i++){
    if(i<FALCrank)
        MPI_Isend((FALCsendbuff[i].vid), FALCsendsize[i], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    if(i>FALCrank)
        MPI_Isend((FALCsendbuff[i-1].vid), FALCsendsize[i-1], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);
}
for(int i=0;i<FALCsize;i++){
    if(i<FALCrank)
        MPI_Recv(FALCrecvbuff[i].vid,graph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    if(i>FALCrank)
    MPI_Recv(FALCrecvbuff[i-1].vid,graph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
}

FALCmsgno++;
for(int i=0;i<FALCsize;i++){
    if(i<FALCrank)
        MPI_Isend((FALCsendbuff[i].dist), FALCsendsize[i], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    if(i>FALCrank)
        MPI_Isend((FALCsendbuff[i-1].dist), FALCsendsize[i-1], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);
}
for(int i=0;i<FALCsize;i++){
    if(i<FALCrank)
        MPI_Recv(FALCrecvbuff[i].dist,graph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i]);
    if(i>FALCrank)
        MPI_Recv(FALCrecvbuff[i-1].dist,graph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i-1]);
}
//changed should be synchronized as it is a global var

FALCmsgno++;
if(FALCrank!=0)for(int i=1;i< FALCsize;i++)MPI_Isend(&changed,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
if(FALCrank==0){
    int tempchanged=0;
    for(int i=1;i<FALCsize;i++){
        MPI_Recv(&tempchanged,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        changed+=tempchanged;
    }

FALCmsgno++;
    for(int i=1;i< FALCsize;i++)MPI_Isend(&changed,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
else {
    
FALCmsgno++;
    MPI_Recv(&changed,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
}
for(int kk=0;kk<(FALCsize-1);kk++){
    MPI_Get_count(&FALCstatus[kk], MPI_INT, &FALCnamount);
    #pragma omp parallel for   num_threads(32)
    for(int i=0;i<FALCnamount;i++){ 
        int vertex= FALCrecvbuff[kk].vid[i];
        if( ( ( struct struct_hgraph  * )(graph.extra))->dist[vertex] > FALCrecvbuff[kk].dist[i])
            ( ( struct struct_hgraph  * )(graph.extra))->dist[vertex] = FALCrecvbuff[kk].dist[i];
    }
}
//here only master node of a point has updated value, sync it over all nodes needed. future work
for(int i=0;i<FALCsize;i++)FALCsendsize[i]=0;

 /*TE=2*///GPU ASS 
//val=2 
//Dtype -1 -1=
if(hipMemcpyFromSymbol(&(hchanged),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3"); 


 if( hchanged==0 )break;

 }//end

 /*TE=2*///GPU ASS 
//val=2 
struct struct_hgraph  temp2;/*xx*/
hipSetDevice(0);
hipMemcpy(&temp2,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy((((struct struct_hgraph  *)(hgraph.extra))->dist),(temp2.dist),sizeof(int)*hgraph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 4"); 


 for (int   i =0;i<hgraph.npoints;i++)printf("%d\n",/*xx*//***/((struct struct_hgraph  *)(hgraph.extra))->dist[i]);

 return ;

 MPI_Finalize();
}//end fun 0 
