#include "hip/hip_runtime.h"

 #include "cpusssp.h"
// checklibfub MIN 
// LIBFUN MIN 
//libfunction call MIN
// clustercomm.c LIB FUNCTION MIN 3rd ARG changed to be communicated
__device__ int   changed =0;
;
 int   hchanged =0;


  int   fchanged =0, fhchanged =0;


 __global__ void   relaxgraph ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints&& ((struct struct_hgraph  *)(graph.extra))->updated[id]==true ){

 ((struct struct_hgraph  *)(graph.extra))->updated[id]=false; 

 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
GMIN(&(((struct struct_hgraph  *)(graph.extra))->dist/*xx*/[ut1]),((struct struct_hgraph  *)(graph.extra))->dist[id]+ut2,/*xx*/changed);//rhs not null


 }//foreach

 }//end fun 0 

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 ((struct struct_hgraph  *)(graph.extra))->olddist[id]=1234567890; 

 ((struct struct_hgraph  *)(graph.extra))->updated[id]=false; 

 }//end fun 0 

}
__global__ void   reset1 ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

 if( ((struct struct_hgraph  *)(graph.extra))->dist[id]<((struct struct_hgraph  *)(graph.extra))->olddist[id] )((struct struct_hgraph  *)(graph.extra))->updated[id]=true; 

 ((struct struct_hgraph  *)(graph.extra))->olddist[id]=((struct struct_hgraph  *)(graph.extra))->dist[id]; 

 }//end fun 0 

}
void   SSSP ( char    *  name ) 
 {

 GGraph  hgraph ;


 /*TE=2*/

 /*TE=2*/

 /*TE=2*/

 /*TE=2*/hgraph.read2(name);

int TPB0=findthreadsperblock(prop0);

int hgraphpointkernelblocks=findblocksize(hgraph,hgraph.npoints,TPB0);
; 

 /*SINGLE FLAG reset 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.npoints;kk+=hgraphpointkernelblocks*TPB0){ 
reset<<<hgraphpointkernelblocks,TPB0>>>(hgraph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
hipDeviceSynchronize();
hipSetDevice(0);


 /*TE=1*///GPU ASS 
bool  falcvt1;
falcvt1=true;
struct struct_hgraph  temp0;
 hipMemcpy(&temp0,((struct struct_hgraph  *)(hgraph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp0.updated[0]),&(falcvt1),sizeof(bool),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 6"); 


 /*TE=1*///GPU ASS 
int   falcvt2;
falcvt2=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(hgraph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist[0]),&(falcvt2),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 7"); 


 
 while(1)  { 

 /*TE=1*///GPU ASS 
int   falcvt3;
falcvt3=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt3),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 8");//val=1 
 


 /*SINGLE FLAG relaxgraph 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.npoints;kk+=hgraphpointkernelblocks*TPB0){ 
relaxgraph<<<hgraphpointkernelblocks,TPB0>>>(hgraph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
hipDeviceSynchronize();
hipSetDevice(0);


 
 //GPU IF STMT 
int   falcvt4;
if(hipMemcpyFromSymbol(&(falcvt4),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 9");
if(falcvt4==0)break;

 /*SINGLE FLAG reset1 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.npoints;kk+=hgraphpointkernelblocks*TPB0){ 
reset1<<<hgraphpointkernelblocks,TPB0>>>(hgraph,kk);
}
hipDeviceSynchronize();
hipSetDevice(0);
hipDeviceSynchronize();
hipSetDevice(0);


 }//end

 struct struct_hgraph   temp2;
 hipMemcpy((void *)&temp2,hgraph.extra,sizeof( struct struct_hgraph ),hipMemcpyDeviceToHost);
struct struct_hgraph   temp3;
 hipMemcpy((void *)&temp3,hgraph.extra,sizeof( struct struct_hgraph ),hipMemcpyDeviceToHost);
int  *temp5=(int  *) malloc(sizeof(int )*hgraph.npoints);
struct struct_hgraph   temp6;
 hipMemcpy((void *)&temp6,hgraph.extra,sizeof(struct struct_hgraph  ),hipMemcpyDeviceToHost);
hipMemcpy(temp5, temp6.dist,sizeof(int )*hgraph.npoints,hipMemcpyDeviceToHost);
for (int   i =0;i<hgraph.npoints;i++) {

 printf("%d\n",/*xx*/temp5[i]);//rhs not null


 }//endfor

 return ;

 }//end fun 0 
int   main ( int   argc ,char    *  argv [ ] ) 
 {
hipGetDeviceProperties(&prop0,0); 
alloc_sync_array();//1


 SSSP(argv[1]);//rhs not null


 }//end fun 0 
