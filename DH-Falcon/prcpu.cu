#include "hip/hip_runtime.h"

 #include "prcpu.h"
// checklibfub ADD 
// LIBFUN ADD 
//libfunction call ADD
void FALCmpiinit(int argc,char **argv){
MPI_Init(&argc,&argv);
MPI_Comm_rank(MPI_COMM_WORLD, &FALCrank);
MPI_Comm_size(MPI_COMM_WORLD, &FALCsize);
  gethostname(FALChostname,255);
hipSetDevice(FALCrank);
hipMalloc(&FALCsendbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCrecvbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCsendsize,sizeof(int)*FALCsize);
hipMalloc(&FALCrecvsize,sizeof(int)*FALCsize);
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}
 FALCstatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCrequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
}
 int   changed =0, hchanged =0;


  int   fchanged =0, fhchanged =0;


 __global__ void   relaxgraph ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

  int   outs =graph.index[p+1].ipe - graph.index[p].ipe;


  float   sum =((struct struct_hgraph  *)(graph.extra))->PR[id]/outs;


 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
GADD(&(((struct struct_hgraph  *)(graph.extra))->val/*xx*/[ut1]),sum,/*xx*/changed);//rhs not null


 }//foreach

 }//end fun 0 

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

 ((struct struct_hgraph  *)(graph.extra))->PR[id]=1/graph.npoints; 

 ((struct struct_hgraph  *)(graph.extra))->val[id]=0.000000; 

 }//end fun 0 

}
__global__ void   reset1 ( GGraph  graph ,float   d ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.npoints){

  float   sum =d*((struct struct_hgraph  *)(graph.extra))->val[id]+(1-d)/graph.npoints;


 ((struct struct_hgraph  *)(graph.extra))->PR[id]=sum; 

 ((struct struct_hgraph  *)(graph.extra))->val[id]=0.000000; 

 }//end fun 0 

}
void   PageRank ( char    *  name ) 
 {

 GGraph  hgraph ;


  int   cnt =0;


 /*TE=2*/

 /*TE=2*/

 /*TE=2*/hgraph.readPointsN(partitionfile,FALCsize);
hgraph.makeNPartitionsMPI(name,FALCrank,FALCsize);
; 

  float   d =0.850000;


 /*SINGLE FLAG reset 0 */
hipSetDevice(FALCrank);
 
reset<<<hgraph.npoints/TPB0+1,TPB0>>>(hgraph,kk);

hipDeviceSynchronize();
hipSetDevice(FALCrank);


 
 while(cnt<100)  { 

 changed=0; 

 sendprefix<<<(hgraph.localpoints+hgraph.remotepoints)/1024+1,1024>>>(hgraph,tempval);
hipDeviceSynchronize();
/*SINGLE FLAG relaxgraph 0 */
hipSetDevice(FALCrank);
 
relaxgraph<<<hgraph.localpoints/TPB0+1,TPB0>>>(hgraph,kk);

hipDeviceSynchronize();
hipSetDevice(FALCrank);
for(int kk=1;kk<FALCsize;kk++){
    int offstart,offend;
    offstart=hgraph.offset[kk-1];
    offend=hgraph.offset[kk];
sendbuff<<<(offend-offstart)/1024+1,1024>>>(hgraph,FALCsendsize,FALCsendbuff,tempval,kk-1,offstart,(offend-offstart));
}
hipDeviceSynchronize();
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i-1]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);

    }}for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&FALCrecvbuff[i],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&FALCrecvbuff[i-1],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    }}//EXTRA DATATYPE val
//1 7 

FALCmsgno++;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.val), temp1, MPI_FLOAT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&FALCsendsize[i-1],sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.val), temp1, MPI_FLOAT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);
    }}for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCrecvbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.val,hgraph.npoints, MPI_FLOAT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCrecvbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.val,hgraph.npoints, MPI_FLOAT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i-1]);
    }
}
for(int kk=0;kk<(FALCsize-1);kk++){
    MPI_Get_count(&FALCstatus[kk], MPI_INT, &FALCnamount);
    update<<< FALCnamount/1024+1,1024>>>(hgraph,FALCrecvbuff,FALCnamount,kk);
}
hipDeviceSynchronize();
//here only master node of a point has updated value, sync it over all nodes needed. future work
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}


 if( cnt<100 )break;

 /*SINGLE FLAG reset1 0 */
hipSetDevice(FALCrank);
 
reset1<<<hgraph.localpoints/TPB0+1,TPB0>>>(hgraph,d,kk);

hipDeviceSynchronize();
hipSetDevice(FALCrank);


 cnt++; 

 }//end

 struct struct_hgraph   temp0;
 hipMemcpy((void *)&temp0,hgraph.extra,sizeof( struct struct_hgraph ),hipMemcpyDeviceToHost);
struct struct_hgraph   temp1;
 hipMemcpy((void *)&temp1,hgraph.extra,sizeof( struct struct_hgraph ),hipMemcpyDeviceToHost);
float  *temp3=(float  *) malloc(sizeof(float )*hgraph.npoints);
struct struct_hgraph   temp4;
 hipMemcpy((void *)&temp4,hgraph.extra,sizeof(struct struct_hgraph  ),hipMemcpyDeviceToHost);
hipMemcpy(temp3, temp4.PR,sizeof(float )*hgraph.npoints,hipMemcpyDeviceToHost);
for (int   i =0;i<hgraph.npoints;i++) {

 printf("%f\n",/*xx*/temp3[i]);//rhs not null


 }//endfor

 return ;

 }//end fun 0 
int   main ( int   argc ,char    *  argv [ ] ) 
 {FALCmpiinit(argc,argv);
sprintf(partitionfile,"%s",argv[2]);


 PageRank(argv[1]);//rhs not null


 MPI_Finalize();
}//end fun 0 
