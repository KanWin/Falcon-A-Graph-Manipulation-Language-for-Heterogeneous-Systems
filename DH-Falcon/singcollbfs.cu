#include "hip/hip_runtime.h"

 #include "singcollbfs.h"
hipDeviceProp_t prop0;
// cluster comm changed 
// cluster comm changed 
void FALCmpiinit(int argc,char **argv){
MPI_Init(&argc,&argv);
MPI_Comm_rank(MPI_COMM_WORLD, &FALCrank);
MPI_Comm_size(MPI_COMM_WORLD, &FALCsize);
FALCsize*=2;
  gethostname(FALChostname,255);
FALCsendbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
FALCrecvbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCgpusendbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCgpurecvbuff,sizeof(struct FALCbuffer )*FALCsize);
FALCsendsize=(int *)malloc(sizeof(int)*FALCsize);
FALCrecvsize=(int *)malloc(sizeof(int)*FALCsize);
hipMalloc(&FALCgpusendsize,sizeof(int)*FALCsize);
hipMalloc(&FALCgpurecvsize,sizeof(int)*FALCsize);
for(int i=0;i<FALCsize;i++){
int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);
FALCsendsize[i]=FALCrecvsize[i]=0;}
 FALCcpustatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCcpurequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
 FALCgpustatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCgpurequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
}
__device__ int   changed =0,GPU_changed, coll1cnt =0,GPU_coll1cnt, coll2cnt =0,GPU_coll2cnt,GPU_hchanged;
;


 __global__ void   relaxgraph ( GGraph  graph ,GColl coll1,GColl coll2,int   val ,int FALCX) 
 {
int id;
 if(threadIdx.x+blockDim.x*blockIdx.x>=coll1.size)return; 
id=coll1.ptr[threadIdx.x+blockDim.x*blockIdx.x];
if( id < graph.localpoints){

 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
if( ((struct struct_hgraph  *)(graph.extra))->dist/*here1*//*xx*/[ut1]>(val/*here1*/+1) ){

 ((struct struct_hgraph  *)(graph.extra))->dist/*here1*//*xx*/[ut1]=val/*here1*/+1; 

 int falcvt1;

falcvt1=GADD(&(coll2.size[0]),1);
coll2.ptr[falcvt1]=t/*here1*/; 

 changed/*here1*/=1; 

 }//end

 }//foreach

 }//end fun 0 

}
__global__ void   reset ( GGraph  graph ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.localpoints+graph.remotepoints){

 ((struct struct_hgraph  *)(graph.extra))->dist[id]=1234567890; 

 }//end fun 0 

}
void   SSSP ( char    *  name ) 
 {

 /*s1 0 0*/HGraph  hgraph ;

 

 GGraph graph;


 

 hgraph.readPointsN(partitionfile,FALCsize);
hgraph.makeNPartitionsMPI(name/*here1*/,2*FALCrank,FALCsize);
hgraph.gpupart.readPointsN(partitionfile,FALCsize);
hgraph.gpupart.makeNPartitionsMPI(name/*here1*/,2*FALCrank+1,FALCsize);
 FALCallocbuff(FALCsendbuff,FALCsize,hgraph.remotepoints);
 FALCallocbuff(FALCrecvbuff,FALCsize,hgraph.npoints);
 FALCgpuallocbuff(FALCgpusendbuff,FALCsize,hgraph.gpupart.remotepoints);
 FALCgpuallocbuff(FALCgpurecvbuff,FALCsize,hgraph.gpupart.npoints);
int TPB0=1024;
alloc_extra_hgraph(hgraph.gpupart,0);
int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 /*TE=1*///GPU ASS 
hgraph/*here1*/.cloneGPU(graph/*here1*/,0 );
int graph/*here1*/flag=0;

alloc_extra_graph(graph,graphflag,graph.npoints);

int TPB0=findthreadsperblock(&prop0);

int graphpointkernelblocks=findblocksize(graph,graph.npoints,TPB0);

int graphedgekernelblocks=findblocksize(graph,graph.nedges,TPB0);
copygraph/*here1*/currentsize(graph/*here1*/);
hipSetDevice(0);
//val=1 
coll2.ptr[falcvt1]=t/*here1*/graph/*here1*/=hgraph/*here1*/; 


 thrust::device_vector<int> coll1temp(graph.npoints);
GColl coll1;
  coll1.ptr=thrust::raw_pointer_cast(&coll1temp[0]);
alloccoll1size(coll1);
thrust::device_vector<int> coll2temp(graph.npoints);
GColl coll2;
  coll2.ptr=thrust::raw_pointer_cast(&coll2temp[0]);
alloccoll2size(coll2);
thrust::device_vector<int> coll3temp(graph.npoints);
GColl coll3;
  coll3.ptr=thrust::raw_pointer_cast(&coll3temp[0]);
alloccoll3size(coll3);


 /*TE=1*///GPU ASS 
int   falcvt2;
falcvt2=0;
struct struct_hgraph  temp1;
 hipMemcpy(&temp1,((struct struct_hgraph  *)(graph.extra)),sizeof(struct struct_hgraph ),hipMemcpyDeviceToHost);
if(hipMemcpy(&(temp1.dist/*here1*/[0]),&(falcvt2),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");/***/((struct struct_hgraph  *)(graph.extra))->dist/*here1*/[0]=0; 


 /*XX3*/((struct struct_hgraph  *)(hgraph.hostparts[0].extra))->dist/*here1*/[0]=0; 

 hipSetDevice(0);
 
reset<<<graph.gpupart.npoints/TPB0+1,TPB0>>>(graph,0);


 /*TE=2*/int falcvt3;
hipMemcpy(&falcvt3,&(coll1.size[0]),sizeof(int),hipMemcpyDeviceToHost);
coll1temp[ falcvt3++]=0;
hipMemcpy(&(coll1.size[0]),&falcvt3,sizeof(int),hipMemcpyHostToDevice); 

 int falcvt4;
hipMemcpy(&falcvt4,&(coll1.size[0]),sizeof(int),hipMemcpyDeviceToHost);
relaxgraph<<<(falcvt4)/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,coll1,coll2,xx,0);
hipDeviceSynchronize();


 
 while(1)  { 

 /*TE=1*///GPU ASS 
int   falcvt5;
falcvt5=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(GPU_changed/*here1*/),&(falcvt5),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 2");//val=1 
changed/*here1*/=0; 


 /*TE=3*///GPU ASS 
//val=3 
coll3.ptr=coll1.ptr;
coll3/*here1*/=coll1/*here1*/; 


 /*TE=3*///GPU ASS 
//val=3 
coll1.ptr=coll2.ptr;
coll1/*here1*/=coll2/*here1*/; 


 /*TE=3*///GPU ASS 
//val=3 
coll2.ptr=coll3.ptr;
coll2/*here1*/=coll3/*here1*/; 


 /*TE=2*///GPU ASS 
//val=2 
//Dtype -1 4=
hipMemcpy(&temp/*here1*/,coll2.size/**/,sizeof(int),hipMemcpyDeviceToHost);
temp/*here1*/=coll2.size/**/; 


 /*TE=1*///GPU ASS 
hipMemcpy(coll1.size/**/,&temp/*here1*/,sizeof(int),hipMemcpyHostToDevice);
coll1.size/**/=temp/*here1*/; 


 temp/*here1*/=0; 

 /*TE=1*///GPU ASS 
hipMemcpy(coll2.size/**/,&temp/*here1*/,sizeof(int),hipMemcpyHostToDevice);
coll2.size/**/=temp/*here1*/; 


 xx/*here1*/++; 

 int falcvt6;
hipMemcpy(&falcvt6,&(coll1.size[0]),sizeof(int),hipMemcpyDeviceToHost);
relaxgraph<<<(falcvt6)/prop0.maxThreadsPerBlock+1,prop0.maxThreadsPerBlock>>>(graph,coll1,coll2,xx,0);
hipDeviceSynchronize();


 
 //GPU IF STMT 
int   falcvt7;
if(hipMemcpyFromSymbol(&(falcvt7),changed/*here1*/,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 3");
if(falcvt7==0)break;

 }//end

 }//end fun 0 
int   main ( int   argc ,char    *  argv [ ] ) 
 {FALCmpiinit(argc,argv);
sprintf(partitionfile,"%s",argv[2]);


 SSSP(argv[1]);//rhs not null


 MPI_Finalize();
}//end fun 0 
TARGET= 5 
