
 #include "kcore.h"
// checklibfub ADD 
// LIBFUN ADD 
//libfunction call ADD
void FALCmpiinit(int argc,char **argv){
MPI_Init(&argc,&argv);
MPI_Comm_rank(MPI_COMM_WORLD, &FALCrank);
MPI_Comm_size(MPI_COMM_WORLD, &FALCsize);
FALCsize*=2;
  gethostname(FALChostname,255);
FALCsendbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
FALCrecvbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCgpusendbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCgpurecvbuff,sizeof(struct FALCbuffer )*FALCsize);
FALCsendsize=(int *)malloc(sizeof(int)*FALCsize);
FALCrecvsize=(int *)malloc(sizeof(int)*FALCsize);
hipMalloc(&FALCgpusendsize,sizeof(int)*FALCsize);
hipMalloc(&FALCgpurecvsize,sizeof(int)*FALCsize);
for(int i=0;i<FALCsize;i++){
int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);
FALCsendsize[i]=FALCrecvsize[i]=0;}
 FALCcpustatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCcpurequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
 FALCgpustatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCgpurequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
}


 void   relaxgraph1 ( int & p , /**u1**/HGraph & graph ,int   level ) 
 {

 ((struct struct_hgraph  *)(graph.extra))->dist1[p]=false; 

 int falcft0=graph.index[p+1]-graph.index[p];
int falcft1=graph.index[p];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
((struct struct_hgraph  *)(graph.extra))->dist1/*xx*/[ut1]=false; 

 }//foreach

 }//end fun 0 
void   relaxgraph ( int & p , /**u1**/HGraph & graph ,int   level ) 
 {
 if( ((struct struct_hgraph  *)(graph.extra))->updated[p]==1 ){

 int falcft3=graph.index[p+1]-graph.index[p];
int falcft4=graph.index[p];
/*XX*/for(int falcft5=0;falcft5<falcft3;falcft5++){
int ut3=2*(falcft4+falcft5);
 int ut4=graph.edges[ut3].ipe;
int ut5=graph.edges[ut3+1].ipe;
HADD(&(((struct struct_hgraph  *)(graph.extra))->dist1/*xx*/[ut4]),1,/*xx*/changed);//rhs not null


 }//foreach

 }//end fun 0 

}
void   reset ( int & t , /**u1**/HGraph & graph ,int   level ) 
 {

 ((struct struct_hgraph  *)(graph.extra))->dist[t]=graph.index[t+1].ipe - graph.index[t].ipe; 

 ((struct struct_hgraph  *)(graph.extra))->updated[t]=0; 

 if( ((struct struct_hgraph  *)(graph.extra))->dist[t]<level )((struct struct_hgraph  *)(graph.extra))->updated[t]=1; 

 }//end fun 0 
void   reset1 ( int & t , /**u1**/HGraph & graph ,int   level ) 
 {

 ((struct struct_hgraph  *)(graph.extra))->dist[t]=((struct struct_hgraph  *)(graph.extra))->dist[t]-((struct struct_hgraph  *)(graph.extra))->dist1[t]; 

 if( ((struct struct_hgraph  *)(graph.extra))->dist[t]<level&&((struct struct_hgraph  *)(graph.extra))->updated[t]==0 )((struct struct_hgraph  *)(graph.extra))->updated[t]=1; 

 if( ((struct struct_hgraph  *)(graph.extra))->updated[t]==1 )((struct struct_hgraph  *)(graph.extra))->updated[t]==2; 

 }//end fun 0 
void   SSSP ( char    *  name ) 
 {

 /*s1 0 0*/HGraph  hgraph ;

 

 

 

 hgraph.readPointsN(partitionfile,FALCsize);
hgraph.makeNPartitionsMPI(name,2*FALCrank,FALCsize);
hgraph.gpupart.readPointsN(partitionfile,FALCsize);
hgraph.gpupart.makeNPartitionsMPI(name,2*FALCrank+1,FALCsize);
 FALCallocbuff(FALCsendbuff,FALCsize,hgraph.remotepoints);
 FALCallocbuff(FALCrecvbuff,FALCsize,hgraph.npoints);
 FALCgpuallocbuff(FALCgpusendbuff,FALCsize,hgraph.gpupart.remotepoints);
 FALCgpuallocbuff(FALCgpurecvbuff,FALCsize,hgraph.gpupart.npoints);
int TPB0=1024;
alloc_extra_hgraph(hgraph.gpupart,0);
int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 /*SINGLE FLAG reset 0 */
hipSetDevice(0);
 
reset<<<hgraph.gpupart.npoints/TPB0+1,TPB0>>>(hgraph,0);
/*SINGLE FLAG reset 0 */
#pragma omp parallel for   num_threads(12)
for(int i=0;i<hgraph.localpoints+hgraph.remotepoints;i++)reset(i,hgraph);

}
}


 
 while(lev<100)  { 

 /*TE=1*///GPU ASS 
int   falcvt1;
falcvt1=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(GPU_changed),&(falcvt1),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 0");//val=1 
changed=0; 


 sendprefix<<<(hgraph.localpoints+hgraph.remotepoints)/1024+1,1024>>>(hgraph,tempgpudist1);
hipDeviceSynchronize();
#pragma omp parallel for num_threads(32)
for(int i=0;i<hgraph.localpoints+hgraph.remotepoints;i++){
    tempdist1[i]= (( struct struct_hgraph  *)(hgraph.extra))->dist1[i];
}
/*SINGLE FLAG relaxgraph 0 */
hipSetDevice(0);
 
relaxgraph<<<hgraph.gpupart.localpoints/TPB0+1,TPB0>>>(hgraph,0);
/*SINGLE FLAG relaxgraph 0 */
#pragma omp parallel for   num_threads(12)
for(int i=0;i<hgraph.localpoints;i++)relaxgraph(i,hgraph);

}
}
for(int kk=1;kk<FALCsize;kk++){
    int offstart,offend;
    offstart=hgraph.gpupart.offset[kk-1];
    offend=hgraph.gpupart.offset[kk];
sendbuff<<<(offend-offstart)/1024+1,1024>>>(hgraph.gpupart,FALCgpusendsize,FALCgpusendbuff,tempgpudist1,kk-1,offstart,(offend-offstart));
hipDeviceSynchronize();
    #pragma omp parallel for   num_threads(32)
    for(int i=hgraph.offset[kk]-1;i<hgraph.offset[kk];i++){
        sendbuff(i,hgraph,FALCsendsize,FALCsendbuff,tempdist1,kk-1);
}
    }
FALcmsgno=0;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i==2*FALCrank|| i==2*FALCrank+1){
    int temp1=0;
    if(i==2*FALCrank+1){
    hipMemcpy( &temp1,&(FALCgpusendsize[i-1]),sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy( &temp,&(FALCgpusendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
    MPI_Isend((temp.vid), temp1, MPI_INT, i/2 ,FALCmsgno, MPI_COMM_WORLD,&FALCgpurequest[i]);//msgno send GPU value same node.
    FALCmsgno++;
}
else{//send from CPU to GPU of same node.rank==0, i==0. rank==1,i==2
     MPI_Isend(FALCsendbuff[i].vid, FALCsendsize[i], MPI_INT, i/2 ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    FALCmsgno++;
}
}
    if(i<FALCrank){
MPI_Isend(FALCsendbuff[i].vid, FALCsendsize[i], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i+1]);
    FALCmsgno++;
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCgpurequest[i]);
    FALCmsgno++;
    }
    if(i>FALCrank){
MPI_Isend(FALCsendbuff[i-1].vid, FALCsendsize[i-1], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    FaLCmsgno++;
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i-1]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCgpurequest[i]);
    FALCmsgno++;
    }
}
int cnt=0,gpucnt=0;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i==2*FALCrank|| i==2*FALCrank+1){
    int temp1=0;
    if(i==2*FALCrank+1){
    for(int kk=0;kk<FALCsize/2;kk++){
    if(kk<FALCrank){
    int kk1=4*(FALCrank-kk);//4
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,kk,kk1+1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
    }
    if(i>FALCrank){
    int kk1=2*(kk-FALCrank);//2
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,kk,kk1+1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
    }
    if(kk==FALCrank){
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,kk, 4*FALCrank, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
    }}
}
else {
    for(int kk=0;kk<FALCsize/2;kk++){
    if(kk<FALCrank){
    int kk1=2*(FALCrank-kk);
    MPI_Recv(FALCrecvbuff[cnt].vid,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
    MPI_Recv(FALCrecvbuff[cnt].vid,hgraph.npoints, MPI_INT,kk, kk1+1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
}
if(kk >FALCrank){
    int kk1=2*(kk-FALCrank)-2;
    MPI_Recv(FALCrecvbuff[cnt].vid,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
    MPI_Recv(FALCrecvbuff[cnt].vid,hgraph.npoints, MPI_INT,kk, kk1+1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
}
if(kk==FALCrank){
    MPI_Recv(FALCrecvbuff[cnt].vid,hgraph.npoints, MPI_INT,kk, 4*FALCrank+1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
}
}
}
}
}
FALCmsgno=0;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i==2*FALCrank|| i==2*FALCrank+1){
    int temp1=0;
    if(i==2*FALCrank+1){
    hipMemcpy( &temp1,&(FALCgpusendsize[i-1]),sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy( &temp,&(FALCgpusendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
    MPI_Isend((temp.dist1), temp1, MPI_INT, i/2 ,FALCmsgno, MPI_COMM_WORLD,&FALCgpurequest[i]);//msgno send GPU value same node.
    FALCmsgno++;
}
else{//send from CPU to GPU of same node.rank==0, i==0. rank==1,i==2
     MPI_Isend(FALCsendbuff[i].dist1, FALCsendsize[i], MPI_INT, i/2 ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    FALCmsgno++;
}
}
    if(i<FALCrank){
MPI_Isend(FALCsendbuff[i].dist1, FALCsendsize[i], MPI_INT, i ,FALCmsgn, MPI_COMM_WORLD,&FALCrequest[i+1]);
    FALCmsgno++;
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.dist1), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCgpurequest[i]);
    FALCmsgno++;
    }
    if(i>FALCrank){
MPI_Isend(FALCsendbuff[i-1].dist1, FALCsendsize[i-1], MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    FaLCmsgno++;
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i-1]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.dist1), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCgpurequest[i]);

    FALCmsgno++;
    }
}
 cnt=0,gpucnt=0;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i==2*FALCrank|| i==2*FALCrank+1){
    int temp1=0;
    if(i==2*FALCrank+1){
    for(int kk=0;kk<FALCsize/2;kk++){
    if(kk<FALCrank){
    int kk1=4*(FALCrank-kk);//4
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.dist1,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.dist1,hgraph.npoints, MPI_INT,kk,kk1+1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
    }
    if(i>FALCrank){
    int kk1=2*(kk-FALCrank);//2
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.dist1,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.dist1,hgraph.npoints, MPI_INT,kk,kk1+1, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
    }
    if(kk==FALCrank){
        hipMemcpy( &temp,&FALCgpurecvbuff[gpucnt],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.dist1,hgraph.npoints, MPI_INT,kk, 4*FALCrank, MPI_COMM_WORLD,&FALCgpustatus[gpucnt]);
    gpucnt++;
    }}
}
else {
    for(int kk=0;kk<FALCsize/2;kk++){
    if(kk<FALCrank){
    int kk1=2*(FALCrank-kk);
    MPI_Recv(FALCrecvbuff[cnt].dist1,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
    MPI_Recv(FALCrecvbuff[cnt].dist1,hgraph.npoints, MPI_INT,kk, kk1+1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
}
if(kk >FALCrank){
    int kk1=2*(kk-FALCrank)-2;
    MPI_Recv(FALCrecvbuff[cnt].dist1,hgraph.npoints, MPI_INT,kk, kk1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
    MPI_Recv(FALCrecvbuff[cnt].dist1,hgraph.npoints, MPI_INT,kk, kk1+1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
}
if(kk==FALCrank){
    MPI_Recv(FALCrecvbuff[cnt].dist1,hgraph.npoints, MPI_INT,kk, 4*FALCrank+1, MPI_COMM_WORLD,&FALCstatus[cnt]);//msgno recv GPU Value
    cnt++;
}
}
}
}
}

for(int kk=0;kk<(FALCsize-1);kk++){
    MPI_Get_count(&FALCstatus[kk], MPI_INT, &FALCnamount);
    update<<< FALCnamount/1024+1,1024>>>(hgraph,FALCrecvbuff,FALCnamount,kk);
}
hipDeviceSynchronize();
//here only master node of a point has updated value, sync it over all nodes needed. future work
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}


 
 //GPU IF STMT 
//GPU_GPU_changed
int   falcvt2;
if(hipMemcpyFromSymbol(&(falcvt2),GPU_changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 1");
if(falcvt2==0&& changed==0)break;

 /*SINGLE FLAG reset1 0 */
hipSetDevice(0);
 
reset1<<<hgraph.gpupart.localpoints/TPB0+1,TPB0>>>(hgraph,0);
/*SINGLE FLAG reset1 0 */
#pragma omp parallel for   num_threads(12)
for(int i=0;i<hgraph.localpoints;i++)reset1(i,hgraph);

}
}


 /*SINGLE FLAG relaxgraph1 0 */
hipSetDevice(0);
 
relaxgraph1<<<hgraph.gpupart.localpoints/TPB0+1,TPB0>>>(hgraph,0);
/*SINGLE FLAG relaxgraph1 0 */
#pragma omp parallel for   num_threads(12)
for(int i=0;i<hgraph.localpoints;i++)relaxgraph1(i,hgraph);

}
}


 /*TE=2*/level++; 

 }//end

 for (int   i =0;i<hgraph.npoints;i++) {

 printf("%d\n",/*xx*//*XX3*/((struct struct_hgraph  *)(hgraph.hostparts[0].extra))->dist[i]);//rhs not null


 }//endfor

 return ;

 }//end fun 0 
int   main ( int   argc ,char    *  argv [ ] ) 
 {FALCmpiinit(argc,argv);
sprintf(partitionfile,"%s",argv[2]);


 /*TE=1*///GPU ASS 
//val=1 
graph.index[t+1].ipe - graph.index[t].ipelevel=atoi(argv[2]); 


 SSSP(argv[1]);//rhs not null


 MPI_Finalize();
}//end fun 0 
