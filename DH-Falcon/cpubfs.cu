
 #include "cpubfs.h"
// cluster comm changed 
void FALCmpiinit(int argc,char **argv){
MPI_Init(&argc,&argv);
MPI_Comm_rank(MPI_COMM_WORLD, &FALCrank);
MPI_Comm_size(MPI_COMM_WORLD, &FALCsize);
FALCsize*=2;
  gethostname(FALChostname,255);
FALCsendbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
FALCrecvbuff=(struct FALCbuffer *)malloc(sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCgpusendbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCgpurecvbuff,sizeof(struct FALCbuffer )*FALCsize);
FALCsendsize=(int *)malloc(sizeof(int)*FALCsize);
FALCrecvsize=(int *)malloc(sizeof(int)*FALCsize);
hipMalloc(&FALCgpusendsize,sizeof(int)*FALCsize);
hipMalloc(&FALCgpurecvsize,sizeof(int)*FALCsize);
for(int i=0;i<FALCsize;i++){
int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);
FALCsendsize[i]=FALCrecvsize[i]=0;}
 FALCcpustatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCcpurequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
 FALCgpustatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCgpurequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
}


 

 void   reset ( int & t , /**u1**/HGraph & graph ) 
 {

 ((struct struct_hgraph  *)(graph.extra))->dist[t]=1234567890; 

 }//end fun 0 
void   BFS ( int & p , /**u1**/HGraph & graph ,int   lev ) 
 {
 if( ((struct struct_hgraph  *)(graph.extra))->dist[p]==lev ){

 

 int falcft0=graph.index[p+1]-graph.index[p];
int falcft1=graph.index[p];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
if( ((struct struct_hgraph  *)(graph.extra))->dist/*xx*/[ut1]>(((struct struct_hgraph  *)(graph.extra))->dist[p]+1) ){

 ((struct struct_hgraph  *)(graph.extra))->dist/*xx*/[ut1]=((struct struct_hgraph  *)(graph.extra))->dist[p]+1; 

 /*TE=1*///GPU ASS 
int   falcvt1;
falcvt1=1;
if(hipMemcpyToSymbol(HIP_SYMBOL(GPU_changed),&(falcvt1),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 0");//val=1 
changed=1; 


 }//end

 }//foreach

 }//end fun 0 

}
int   main ( int   argc ,char    *  name [ ] ) 
 {FALCmpiinit(argc,argv);
sprintf(partitionfile,"%s",argv[2]);


 /*s1 0 0*/HGraph  hgraph ;

 

 hgraph.readPointsN(partitionfile,FALCsize);
hgraph.makeNPartitionsMPI(name[1],2*FALCrank,FALCsize);
hgraph.gpupart.readPointsN(partitionfile,FALCsize);
hgraph.gpupart.makeNPartitionsMPI(name[1],2*FALCrank+1,FALCsize);
 FALCallocbuff(FALCsendbuff,FALCsize,hgraph.remotepoints);
 FALCallocbuff(FALCrecvbuff,FALCsize,hgraph.npoints);
 FALCgpuallocbuff(FALCgpusendbuff,FALCsize,hgraph.gpupart.remotepoints);
 FALCgpuallocbuff(FALCgpurecvbuff,FALCsize,hgraph.gpupart.npoints);
int TPB0=1024;
alloc_extra_hgraph(hgraph.gpupart,0);
int hosthgraph=0;
alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
; 

 hipSetDevice(0);
 
reset<<<hgraph.gpupart.npoints/TPB0+1,TPB0>>>(hgraph,0);
#pragma omp parallel for   num_threads(12)
for(int i=0;i<hgraph.localpoints+hgraph.remotepoints;i++)reset(i,hgraph);

}
}


 /*XX3*/((struct struct_hgraph  *)(hgraph.hostparts[0].extra))->dist[0]=0; 

 
 while(1)  { 

 /*TE=1*///GPU ASS 
int   falcvt2;
falcvt2=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(GPU_changed),&(falcvt2),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 1");//val=1 
changed=0; 


 hipSetDevice(0);
 
BFS<<<hgraph.gpupart.localpoints/TPB0+1,TPB0>>>(hgraph,lev,0);
#pragma omp parallel for   num_threads(12)
for(int i=0;i<hgraph.localpoints;i++)BFS(i,,lev);

}
}
//changed should be synchronized as it is a global var

FALCmsgno++;
if(FALCrank!=0){
for(int i=1;i< FALCsize;i++){
int temp;
hipMemcpyFromSymbol(&temp,HIP_SYMBOL(GPU_changed),sizeof(int),0,hipMemcpyDeviceToHost);
temp+=changed;
MPI_Isend(&temp,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
}
if(FALCrank==0){
    int tempchanged=0;
    int temp0;
    hipMemcpyFromSymbol(&temp0,HIP_SYMBOL(GPU_changed),sizeof(int),0,hipMemcpyDeviceToHost);
temp0+=changed;
    for(int i=1;i<FALCsize;i++){
        MPI_Recv(&tempchanged,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        temp0+=tempchanged;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(GPU_changed),&temp0,sizeof(int),0,hipMemcpyHostToDevice);
    changed=temp0;

FALCmsgno++;
    for(int i=1;i< FALCsize;i++)MPI_Isend(&temp0,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
else {
    
FALCmsgno++;
    int temp0;
    MPI_Recv(&temp0,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    hipMemcpyToSymbol(HIP_SYMBOL(changed),&temp0,sizeof(int),0,hipMemcpyHostToDevice);
    changed=temp0;
}

for(int kk=0;kk<(FALCsize-1);kk++){
    MPI_Get_count(&FALCstatus[kk], MPI_INT, &FALCnamount);
    update<<< FALCnamount/1024+1,1024>>>(hgraph,FALCrecvbuff,FALCnamount,kk);
}
hipDeviceSynchronize();
//here only master node of a point has updated value, sync it over all nodes needed. future work
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}


 
 //GPU IF STMT 
//GPU_GPU_changed
int   falcvt3;
if(hipMemcpyFromSymbol(&(falcvt3),GPU_changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 2");
if(falcvt3==0&& changed==0)break;

 /*TE=2*/lev++; 

 }//end

 for (int   i =0;i<hgraph.npoints;i++)printf("%d\n",/*xx*//*XX3*/((struct struct_hgraph  *)(hgraph.hostparts[0].extra))->dist[i]);

 return ;

 MPI_Finalize();
}//end fun 0 
