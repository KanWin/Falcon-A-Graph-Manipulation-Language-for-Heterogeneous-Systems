#include "hip/hip_runtime.h"

 #include "singlemst.h"
// cluster comm t1 
// cluster comm t2 
// cluster comm t3 
// cluster comm t4 
// cluster comm changed 
// cluster comm t1 
// cluster comm t2 
// cluster comm t3 
// cluster comm t1 
// cluster comm t2 
// clustercomm.c LIB FUNCTION MIN 3rd ARG changed to be communicated
// clustercomm.c LIB FUNCTION MIN 3rd ARG changed to be communicated
void FALCmpiinit(int argc,char **argv){
MPI_Init(&argc,&argv);
MPI_Comm_rank(MPI_COMM_WORLD, &FALCrank);
MPI_Comm_size(MPI_COMM_WORLD, &FALCsize);
  gethostname(FALChostname,255);
hipMalloc(&FALCsendbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCrecvbuff,sizeof(struct FALCbuffer )*FALCsize);
hipMalloc(&FALCsendsize,sizeof(int)*FALCsize);
hipMalloc(&FALCrecvsize,sizeof(int)*FALCsize);
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}
 FALCstatus=(MPI_Status *)malloc(sizeof(MPI_Status)*FALCsize);
 FALCrequest=(MPI_Request *)malloc(sizeof(MPI_Request)*FALCsize);
}


 __device__ int   changed ;
;
 int   hchanged ;


 __global__ void   reset ( GGraph  graph ,GSet  set ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.localpoints){

 ((struct struct_hgraph  *)(graph.extra))->Weight[id]=99999999; 

 ((struct struct_hgraph  *)(graph.extra))->minedge[id]=99999999; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[id].lock=0; 

 }//end fun 0 

}
__global__ void   minset ( GGraph  graph ,GSet  set ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.localpoints){

  int   ch ;


 int t1;

 int t2;

 int falcft0=graph.index[id+1]-graph.index[id];
int falcft1=graph.index[id];
/*XX*/for(int falcft2=0;falcft2<falcft0;falcft2++){
int ut0=2*(falcft1+falcft2);
 int ut1=graph.edges[ut0].ipe;
int ut2=graph.edges[ut0+1].ipe;
t1=set./**/parent[id]; 

 ((struct struct_hgraph  *)(graph.extra))->minedge[id]=99999999; 

 t2=set./**/parent[ut1]; 

 if( t1!=t2 ){

 GMIN(&(((struct struct_hgraph  *)(graph.extra))->Weight[t1]),ut2,/*xx*/changed);//rhs not null


 GMIN(&(((struct struct_hgraph  *)(graph.extra))->Weight[t2]),ut2,/*xx*/changed);//rhs not null


 }//end

 }//foreach

 }//end fun 0 

}
__global__ void   Minedge ( GGraph  graph ,GSet  set ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.localpoints){

 int t1;

 int t2;

  int   t3 ;


 int e;

 int falcft3=graph.index[id+1]-graph.index[id];
int falcft4=graph.index[id];
/*XX*/for(int falcft5=0;falcft5<falcft3;falcft5++){
int ut3=2*(falcft4+falcft5);
 int ut4=graph.edges[ut3].ipe;
int ut5=graph.edges[ut3+1].ipe;
t1=set./**/parent[id]; 

 t2=set./**/parent[ut4]; 

 t3=struct struct_hgraph  temp0;/*xx*/
hipSetDevice(0);
((struct struct_hgraph  *)(graph.extra))->getWeight(id,/*xx*/ut4); 

 if( t1!=t2 ){

 if( t3==((struct struct_hgraph  *)(graph.extra))->Weight[t1] ){

  int   unnikri =0;


 if(atomicCAS( &(((struct struct_hgraph  *)(graph.extra))->minppty[t1].lock),FALCRANK,MAX_INT-1)==FALCRANK){

 e=ut3/2; 

 ((struct struct_hgraph  *)(graph.extra))->mark[e]=1; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].src=p; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].dst=t; 

 ((struct struct_hgraph  *)(graph.extra))->Weight[t1]=t3; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t1].set=t2; 

 }//end

 }//end

 if( t3==((struct struct_hgraph  *)(graph.extra))->Weight[t2] ){

 if(atomicCAS( &(((struct struct_hgraph  *)(graph.extra))->minppty[t2].lock),FALCRANK,MAX_INT-1)==FALCRANK){

 e=ut3/2; 

 ((struct struct_hgraph  *)(graph.extra))->mark[e]=1; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].src=p; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].dst=t; 

 ((struct struct_hgraph  *)(graph.extra))->Weight[t2]=t3; 

 ((struct struct_hgraph  *)(graph.extra))->minppty[t2].set=t1; 

 }//end

 }//end

 }//end

 }//foreach

 }//end fun 0 

}
__global__ void   mstunion ( GGraph  graph ,GSet  set ,int FALCX) 
 {
/* 0 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.localpoints){

 int t1;

 int t2;

  int   t3 , t4 ;


 t1=set./**/parent[id]; 

 t2=((struct struct_hgraph  *)(graph.extra))->minppty[t1].set; 

 t3=((struct struct_hgraph  *)(graph.extra))->minppty[t1].lock; 

 t4=((struct struct_hgraph  *)(graph.extra))->minppty[t2].lock; 

 if( t1!=t2&&t3==1 ){

 setUnion(t1,/*xx*/t2); 

 changed=1; 

 }//end

 }//end fun 0 

}
__global__ void   initmark ( GGraph  graph ,int FALCX) 
 {
/* 1 xx*/int id= blockIdx.x * blockDim.x + threadIdx.x+FALCX;
if( id < graph.nedges){

 ((struct struct_hgraph  *)(graph.extra))->mark[id]=999999999; 

 }//end fun 0 

}
int   main ( int   argc ,char    *  argv [ ] ) 
 {FALCmpiinit(argc,argv);
sprintf(partitionfile,"%s",argv[2]);


 GGraph  hgraph ;


 /*TE=2*/

 /*TE=2*/

 /*TE=2*/

 /*TE=2*/

 /*TE=2*///better to read graph in a temporary HGraph object and the clone to GGraph.
//Temporary HGraph object can be inside the GGraph class itself.
//For GGraph do not allocate offset in GPU. It is not needed in any kernel.
hgraph.readPointsN(partitionfile,FALCsize);
hgraph.makeNPartitionsMPI(argv[1],FALCrank,FALCsize);
int hgraphflag=0;
alloc_extra_hgraph(hgraph,hgraphflag,hgraph.npoints);
 FALCallocbuff(FALCsendbuff,FALCsize,hgraph.remotepoints);
 FALCallocbuff(FALCrecvbuff,FALCsize,hgraph.npoints);
int TPB0=1024;
; 

 GSet hset;
hset.allocate(hgraph.npoints);

 /*TE=1*///GPU ASS 
int   falcvt1;
falcvt1=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt1),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 7");//val=1 
t1!=t2&&t3==1 


 /*SINGLE FLAG initmark 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.nedges;kk+=hgraphedgekernelblocks*TPB0){ 
initmark<<<hgraphedgekernelblocks,TPB0>>>(hgraph,kk);}
hipDeviceSynchronize();
hipSetDevice(0);


 
 while(1)  { 

 /*TE=1*///GPU ASS 
int   falcvt2;
falcvt2=0;
if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(falcvt2),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 8");//val=1 
 


 /*SINGLE FLAG reset 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.nedges;kk+=hgraphedgekernelblocks*TPB0){ 
reset<<<hgraphedgekernelblocks,TPB0>>>(hipSetDevice(0);
 
reset<<<hgraph.localpoints/TPB0+1,TPB0>>>(hgraph,hset,0);
hipDeviceSynchronize();
hipSetDevice(0);


 sendprefix<<<(hgraph.localpoints+hgraph.remotepoints)/1024+1,1024>>>(hgraph,tempWeight,tempWeight);
hipDeviceSynchronize();
/*SINGLE FLAG minset 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.nedges;kk+=hgraphedgekernelblocks*TPB0){ 
minset<<<hgraphedgekernelblocks,TPB0>>>(hipSetDevice(0);
 
minset<<<hgraph.localpoints/TPB0+1,TPB0>>>(hgraph,hset,0);
hipDeviceSynchronize();
hipSetDevice(0);
for(int kk=1;kk<FALCsize;kk++){
    int offstart,offend;
    offstart=hgraph.offset[kk-1];
    offend=hgraph.offset[kk];
sendbuff<<<(offend-offstart)/1024+1,1024>>>(hgraph,FALCsendsize,FALCsendbuff,tempWeight,tempWeight,kk-1,offstart,(offend-offstart));
}
hipDeviceSynchronize();
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i-1]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);

    }}for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&FALCrecvbuff[i],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&FALCrecvbuff[i-1],sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.vid,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    }}
FALCmsgno++;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.Weight), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&FALCsendsize[i-1],sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.Weight), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);
    }}for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCrecvbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.Weight,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCrecvbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.Weight,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i-1]);
    }
}//changed should be synchronized as it is a global var

FALCmsgno++;
if(FALCrank!=0){
for(int i=1;i< FALCsize;i++){
int temp;
hipMemcpyFromSymbol(&temp,HIP_SYMBOL(changed),sizeof(int),0,hipMemcpyDeviceToHost);
MPI_Isend(&temp,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
}
if(FALCrank==0){
    int tempchanged=0;
    int temp0;
    hipMemcpyFromSymbol(&temp0,HIP_SYMBOL(changed),sizeof(int),0,hipMemcpyDeviceToHost);
    for(int i=1;i<FALCsize;i++){
        MPI_Recv(&tempchanged,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        temp0+=tempchanged;
    hipMemcpyToSymbol(HIP_SYMBOL(changed),&temp0,sizeof(int),0,hipMemcpyHostToDevice);
    }

FALCmsgno++;
    for(int i=1;i< FALCsize;i++)MPI_Isend(&temp0,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
else {
    
FALCmsgno++;
    int temp0;
    MPI_Recv(&temp0,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    hipMemcpyToSymbol(HIP_SYMBOL(changed),&temp0,sizeof(int),0,hipMemcpyHostToDevice);
}

FALCmsgno++;
for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[i]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.Weight), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCsendbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&FALCsendsize[i-1],sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.Weight), temp1, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i-1]);
    }}for(int i=0;i<FALCsize;i++){
    struct FALCbuffer temp;
    if(i<FALCrank){
        hipMemcpy( &temp,&(FALCrecvbuff[i]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.Weight,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i]);
    }    if(i>FALCrank){
        hipMemcpy( &temp,&(FALCrecvbuff[i-1]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        MPI_Recv(temp.Weight,hgraph.npoints, MPI_INT,i, FALCmsgno, MPI_COMM_WORLD,&FALCstatus[i-1]);
    }
}//changed should be synchronized as it is a global var

FALCmsgno++;
if(FALCrank!=0){
for(int i=1;i< FALCsize;i++){
int temp;
hipMemcpyFromSymbol(&temp,HIP_SYMBOL(changed),sizeof(int),0,hipMemcpyDeviceToHost);
MPI_Isend(&temp,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
}
if(FALCrank==0){
    int tempchanged=0;
    int temp0;
    hipMemcpyFromSymbol(&temp0,HIP_SYMBOL(changed),sizeof(int),0,hipMemcpyDeviceToHost);
    for(int i=1;i<FALCsize;i++){
        MPI_Recv(&tempchanged,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        temp0+=tempchanged;
    hipMemcpyToSymbol(HIP_SYMBOL(changed),&temp0,sizeof(int),0,hipMemcpyHostToDevice);
    }

FALCmsgno++;
    for(int i=1;i< FALCsize;i++)MPI_Isend(&temp0,1,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,&FALCrequest[i-1]);
}
else {
    
FALCmsgno++;
    int temp0;
    MPI_Recv(&temp0,1,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
    hipMemcpyToSymbol(HIP_SYMBOL(changed),&temp0,sizeof(int),0,hipMemcpyHostToDevice);
}

for(int kk=0;kk<(FALCsize-1);kk++){
    MPI_Get_count(&FALCstatus[kk], MPI_INT, &FALCnamount);
    update<<< FALCnamount/1024+1,1024>>>(hgraph,FALCrecvbuff,FALCnamount,kk);
}
hipDeviceSynchronize();
//here only master node of a point has updated value, sync it over all nodes needed. future work
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}


 /*SINGLE FLAG Minedge 1 */
for(int kk=1;kk<FALCsize;kk++){
    int offstart,offend;
    offstart=hgraph.offset[kk-1];
    offend=hgraph.offset[kk];
sendbuffsingle<<<(offend-offstart)/1024+1,1024>>>(hgraph,FALCsendsize,FALCsendbuff,templock,kk-1,offstart,(offend-offstart));
}
hipDeviceSynchronize();
if(FALCRANK!=0){
    struct FALCbuffer temp;
        hipMemcpy( &temp,&(FALCsendbuff[0]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        hipMemcpy( &temp1,&(FALCsendsize[0]),sizeof(int),hipMemcpyDeviceToHost);
        MPI_Isend((temp.vid), temp1, MPI_INT,0,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);
        MPI_Isend((temp.lock), temp1, MPI_INT, i ,FALCmsgno+1, MPI_COMM_WORLD,&FALCrequest[i]);
    }
    if(FALCrank==0){
for(int i=1;i<FALCsize;i++){
        hipMemcpy( &temp,&(FALCsendbuff[0]),sizeof(struct FALCbuffer),hipMemcpyDeviceToHost);
        int temp1;
        MPI_Recv((temp.vid), hgraph.npoints, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);

        MPI_Recv((temp.lock), hgraph.npoints, MPI_INT, i ,FALCmsgno, MPI_COMM_WORLD,&FALCrequest[i]);

//now update here
    MPI_Get_count(&FALCstatus[i], MPI_INT, &FALCnamount);
    updatesingle<<< FALCnamount/1024+1,1024>>>(hgraph,FALCrecvbuff,FALCnamount,i);
hipDeviceSynchronize();
    }
struct struct_hgraph  ftt;
hipMemcpy(&ftt,( (struct struct_hgraph  *)(hgraph.extra)),sizeof(struct struct struct_hgraph ),hipMemcpyDeviceToHost);
for(int i=1;i<FALCsize;i++)MPI_Send(temp.lock,hgraph.npoints,MPI_INT,i,FALCmsgno,MPI_COMM_WORLD,&FALCRequest[i]);
}
if(FALCRANK!=0){
struct struct_hgraph  ftt;
hipMemcpy(&ftt,( (struct struct_hgraph  *)(hgraph.extra)),sizeof(struct struct struct_hgraph ),hipMemcpyDeviceToHost);
MPI_Recv(temp.lock,hgraph.npoints,MPI_INT,0,FALCmsgno,MPI_COMM_WORLD,&FALCstatus[0]);
}
hipMemcpy(( (struct struct_hgraph  *)(hgraph.extra)),sizeof(struct struct struct_hgraph ),hipMemcpyHostToDevice);
for(int i=0;i<FALCsize;i++){int temp=0;
hipMemcpy(&FALCsendsize[i],&temp,sizeof(int),hipMemcpyHostToDevice);}
hipSetDevice(0);
for(int kk=0;kk<hgraph.nedges;kk+=hgraphedgekernelblocks*TPB0){ 
Minedge<<<hgraphedgekernelblocks,TPB0>>>(hipSetDevice(0);
 
Minedge<<<hgraph.localpoints/TPB0+1,TPB0>>>(hgraph,hset,0);
hipDeviceSynchronize();
hipSetDevice(0);


 /*SINGLE FLAG mstunion 0 */
hipSetDevice(0);
for(int kk=0;kk<hgraph.nedges;kk+=hgraphedgekernelblocks*TPB0){ 
mstunion<<<hgraphedgekernelblocks,TPB0>>>(hipSetDevice(0);
 
mstunion<<<hgraph.localpoints/TPB0+1,TPB0>>>(hgraph,hset,0);
hipDeviceSynchronize();
hipSetDevice(0);
hipSetDevice(0);
for(int kk=0;ii<hgraph.tot_size;kk+=hgraphpointkernelblocksize*TPB0){
findset<<<hgraphpointkernelblcoksize,TPB0>>>( hset,kk);
}
hipDeviceSynchronize();hipSetDevice(0);


 
 //GPU IF STMT 
int   falcvt3;
if(hipMemcpyFromSymbol(&(falcvt3),changed,sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 9");
if(falcvt3==0)break;

 }//end

  unsigned  int   mst =0;


 
 if( ((struct struct_hgraph  *)(hgraph.extra))->mark[(null)/2]==1 )mst=mst+hgraph.edges[(null)+1].ipe; 
 }//foreach

 printf("mst cost=%lu",/*xx*/mst);//rhs not null


 MPI_Finalize();
}//end fun 0 
